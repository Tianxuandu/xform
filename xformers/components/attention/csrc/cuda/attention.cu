#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/Parallel.h>
#include <torch/library.h>
#include <cmath>
#include <vector>

#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include "sputnik/vector_utils.h"

namespace {

template <typename integer>
constexpr __host__ __device__ inline integer ceil_div(integer n, integer m) {
  return (n + m - 1) / m;
}

template <typename scalar_t>
constexpr __host__ __device__ bool integerIsPowerOf2(scalar_t v) {
  return (v && !(v & (v - 1)));
}

template <typename scalar_t>
__device__ __forceinline__ void iMul(scalar_t x1, float4* out) {
  out[0].x *= x1;
  out[0].y *= x1;
  out[0].z *= x1;
  out[0].w *= x1;
}

template <typename scalar_t>
__device__ __forceinline__ void iMul(scalar_t x1, float2* out) {
  out[0].x *= x1;
  out[0].y *= x1;
}

template <typename scalar_t>
__device__ __forceinline__ void iMul(scalar_t x1, float* out) {
  out[0] *= x1;
}

template <typename scalar_t>
__device__ __forceinline__ void iDiv(scalar_t x1, float4* out) {
  out[0].x /= x1;
  out[0].y /= x1;
  out[0].z /= x1;
  out[0].w /= x1;
}

template <typename scalar_t>
__device__ __forceinline__ void iDiv(scalar_t x1, float2* out) {
  out[0].x /= x1;
  out[0].y /= x1;
}

template <typename scalar_t>
__device__ __forceinline__ void iDiv(scalar_t x1, float* out) {
  out[0] /= x1;
}

template <typename scalar_t, int WARP_SIZE>
__device__ __forceinline__ scalar_t warpSum(scalar_t val) {
  for (int stride = WARP_SIZE / 2; stride > 0; stride >>= 1) {
    val += __shfl_xor_sync(0xffffffff, val, stride, WARP_SIZE);
  }
  return val;
}

template <typename scalar_t, int WARP_SIZE>
__device__ __forceinline__ float2 warpSum(float2 val) {
  for (int stride = WARP_SIZE / 2; stride > 0; stride >>= 1) {
    val.x += __shfl_xor_sync(0xffffffff, val.x, stride, WARP_SIZE);
    val.y += __shfl_xor_sync(0xffffffff, val.y, stride, WARP_SIZE);
  }
  return val;
}

template <typename scalar_t, int WARP_SIZE>
__device__ __forceinline__ float4 warpSum(float4 val) {
  for (int stride = WARP_SIZE / 2; stride > 0; stride >>= 1) {
    val.x += __shfl_xor_sync(0xffffffff, val.x, stride, WARP_SIZE);
    val.y += __shfl_xor_sync(0xffffffff, val.y, stride, WARP_SIZE);
    val.z += __shfl_xor_sync(0xffffffff, val.z, stride, WARP_SIZE);
    val.w += __shfl_xor_sync(0xffffffff, val.w, stride, WARP_SIZE);
  }
  return val;
}

template <typename scalar_t, int WARP_SIZE>
__device__ __forceinline__ scalar_t warpMax(scalar_t val) {
  for (int stride = WARP_SIZE / 2; stride > 0; stride >>= 1) {
    scalar_t tmp = __shfl_xor_sync(0xffffffff, val, stride, WARP_SIZE);
    val = tmp > val ? tmp : val;
  }
  return val;
}

template <typename scalar_t, typename vec_t, int kBlockSizeK, int kBlockSizeQ>
__device__ void compute_dot(
    vec_t* queries[kBlockSizeQ],
    vec_t* keys,
    scalar_t out[kBlockSizeQ][kBlockSizeK],
    int64_t K) {
  constexpr int kVecSize = sizeof(vec_t) / sizeof(scalar_t);
  vec_t q_i[kBlockSizeQ];
  for (int64_t k = 0; k < K / kVecSize; k += 1) {
#pragma unroll
    for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
      q_i[q_item_idx] = __ldg(queries[q_item_idx] + k);
    }
#pragma unroll
    for (int64_t k_item_idx = 0; k_item_idx < kBlockSizeK; k_item_idx++) {
      vec_t k_i = keys[k + K / kVecSize * k_item_idx];
#pragma unroll
      for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
        sputnik::VectorCompute<vec_t>::Dot(
            q_i[q_item_idx], k_i, &out[q_item_idx][k_item_idx]);
      }
    }
  }
}

template <typename scalar_t, typename vec_t, int kBlockSizeK, int kBlockSizeQ, int BUFFER_SIZE>
__device__ void compute_final_mult(
    vec_t* vi,
    scalar_t s_delta[kBlockSizeQ][kBlockSizeK],
    scalar_t m_delta[kBlockSizeQ],
    vec_t buffer[kBlockSizeQ][BUFFER_SIZE] /*TODO fix me*/,
    int64_t K) {
  constexpr int kVecSize = sizeof(vec_t) / sizeof(scalar_t);

  for (int64_t k = 0; k < K / kVecSize; k += 1) {
#pragma unroll
    for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
      iMul<scalar_t>(m_delta[q_item_idx], &buffer[q_item_idx][k]);
    }
#pragma unroll
    for (int64_t k_item_idx = 0; k_item_idx < kBlockSizeK; k_item_idx++) {
      vec_t tmp2 = vi[k + K / kVecSize * k_item_idx];

#pragma unroll
      for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
        sputnik::VectorCompute<vec_t>::FMA(
            s_delta[q_item_idx][k_item_idx], tmp2, &buffer[q_item_idx][k]);
      }
    }
  }
}

template <typename scalar_t, int kBlockSizeK, int kBlockSizeQ>
__device__ __forceinline__ void compute_max(
    scalar_t a[kBlockSizeQ][kBlockSizeK],
    scalar_t b[kBlockSizeQ],
    scalar_t out[kBlockSizeQ]) {
#pragma unroll
  for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
    out[q_item_idx] =
        a[q_item_idx][0] > b[q_item_idx] ? a[q_item_idx][0] : b[q_item_idx];
#pragma unroll
    for (int64_t k_item_idx = 1; k_item_idx < kBlockSizeK; k_item_idx++) {
      out[q_item_idx] = a[q_item_idx][k_item_idx] > out[q_item_idx]
          ? a[q_item_idx][k_item_idx]
          : out[q_item_idx];
    }
  }
}

template <typename scalar_t, int kBlockSizeK, int kBlockSizeQ>
__device__ __forceinline__ void compute_scaling_coeffs(
    scalar_t m_i[kBlockSizeQ],
    scalar_t m_prime[kBlockSizeQ],
    scalar_t si[kBlockSizeQ][kBlockSizeK],
    scalar_t m_delta[kBlockSizeQ],
    scalar_t s_delta[kBlockSizeQ][kBlockSizeK]) {
#pragma unroll
  for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++)
    m_delta[q_item_idx] = std::exp(m_prime[q_item_idx] - m_i[q_item_idx]);
#pragma unroll
  for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++)
#pragma unroll
    for (int64_t k_item_idx = 0; k_item_idx < kBlockSizeK; k_item_idx++)
      s_delta[q_item_idx][k_item_idx] =
          std::exp(si[q_item_idx][k_item_idx] - m_i[q_item_idx]);
}

template <typename scalar_t, int kBlockSizeK, int kBlockSizeQ>
__device__ __forceinline__ void update_scaling_coeffs(
    scalar_t m_delta[kBlockSizeQ],
    scalar_t m_i[kBlockSizeQ],
    scalar_t s_delta[kBlockSizeQ][kBlockSizeK],
    scalar_t m_prime[kBlockSizeQ],
    scalar_t s_prime[kBlockSizeQ]) {
#pragma unroll
  for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
    s_prime[q_item_idx] = s_prime[q_item_idx] * m_delta[q_item_idx];
#pragma unroll
    for (int64_t k_item_idx = 0; k_item_idx < kBlockSizeK; k_item_idx++)
      s_prime[q_item_idx] += s_delta[q_item_idx][k_item_idx];

    m_prime[q_item_idx] = m_i[q_item_idx];
  }
}

template <typename scalar_t, typename vec_t, int kBlockSizeK, int kBlockSizeQ, int BUFFER_SIZE>
__device__ void compute_loop(
    vec_t* query_block[kBlockSizeQ],
    vec_t* key_i,
    vec_t* value_i,
    scalar_t m_prime[kBlockSizeQ],
    scalar_t s_prime[kBlockSizeQ],
    vec_t buffer[kBlockSizeQ][BUFFER_SIZE] /*TODO fix me*/,
    int64_t K) {

    scalar_t si[kBlockSizeQ][kBlockSizeK] = {0};
    compute_dot<scalar_t, vec_t, kBlockSizeK, kBlockSizeQ>(
        query_block, key_i, si, K);

    scalar_t m_i[kBlockSizeQ];
    compute_max<scalar_t, kBlockSizeK, kBlockSizeQ>(si, m_prime, m_i);

    scalar_t m_delta[kBlockSizeQ];
    scalar_t s_delta[kBlockSizeQ][kBlockSizeK];

    compute_scaling_coeffs<scalar_t, kBlockSizeK, kBlockSizeQ>(
        m_i, m_prime, si, m_delta, s_delta);

    compute_final_mult<scalar_t, vec_t, kBlockSizeK, kBlockSizeQ, BUFFER_SIZE>(
        value_i, s_delta, m_delta, buffer, K);

    update_scaling_coeffs<scalar_t, kBlockSizeK, kBlockSizeQ>(
        m_delta, m_i, s_delta, m_prime, s_prime);
}


template <typename scalar_t, typename vec_t, int kBlockSizeQ, int WARP_SIZE, int BUFFER_SIZE>
__device__ __forceinline__ void aggregate_coeffs(
    scalar_t m_prime[kBlockSizeQ],
    scalar_t s_prime[kBlockSizeQ],
    vec_t buffer[kBlockSizeQ][BUFFER_SIZE] /*TODO fix me*/,
    int64_t K) {
  constexpr int kVecSize = sizeof(vec_t) / sizeof(scalar_t);
  for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
    scalar_t m_i = m_prime[q_item_idx];
    scalar_t s_i = s_prime[q_item_idx];
    m_prime[q_item_idx] = warpMax<scalar_t, WARP_SIZE>(m_prime[q_item_idx]);
    scalar_t m_delta = std::exp(m_i - m_prime[q_item_idx]);
    scalar_t s_delta = s_i * m_delta;
    s_delta = warpSum<scalar_t, WARP_SIZE>(s_delta);
    s_prime[q_item_idx] = s_delta;
    for (int64_t k = 0; k < K / kVecSize; k += 1) {
      vec_t tmp = buffer[q_item_idx][k];
      iMul<scalar_t>(m_delta, &tmp);
      tmp = warpSum<vec_t, WARP_SIZE>(tmp);
      buffer[q_item_idx][k] = tmp;
    }
  }
}


template <
    typename scalar_t,
    typename vec_t = float4,
    int kBlockSizeK = 32,
    int kBlockSizeQ = 2,
    int WARP_SIZE = 4,
    int BUFFER_SIZE = 8>
__global__ void attention_kernel(
    at::PackedTensorAccessor<scalar_t, 3> output,
    at::PackedTensorAccessor<scalar_t, 3> query,
    at::PackedTensorAccessor<scalar_t, 3> key,
    at::PackedTensorAccessor<scalar_t, 3> value) {
  constexpr int kVecSize = sizeof(vec_t) / sizeof(scalar_t);
  static_assert(integerIsPowerOf2(kBlockSizeK * WARP_SIZE), "kBlockSizeK * WARP_SIZE should be a power of 2");
  int64_t K = query.size(2);
  int64_t B = query.size(0);
  int64_t M = query.size(1);
  int64_t N = key.size(1);

  int64_t batch_idx = blockIdx.y;
  int64_t query_idx =
      blockIdx.x * (blockDim.y * kBlockSizeQ) + threadIdx.y * kBlockSizeQ;

  if (query_idx >= M)
    return;

  vec_t* query_block[kBlockSizeQ];
  vec_t* output_block[kBlockSizeQ];
  vec_t buffer[kBlockSizeQ][BUFFER_SIZE] = {0}; // TODO == K / 4
  scalar_t s_prime[kBlockSizeQ] = {0};
  scalar_t m_prime[kBlockSizeQ] = {-std::numeric_limits<scalar_t>::infinity()};
  for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
    int64_t index = query_idx + q_item_idx;
    index = index >= M ? M - 1 : index;
    query_block[q_item_idx] = reinterpret_cast<vec_t*>(
        query[batch_idx][index].data());
    output_block[q_item_idx] = reinterpret_cast<vec_t*>(
        output[batch_idx][index].data());
  }

  int64_t l = threadIdx.x * kBlockSizeK;
  constexpr int64_t step = kBlockSizeK * WARP_SIZE;
  // this is equivalent to N - N % step, but faster
  // guaranteed to be the same as step is a power of 2
  int64_t end_iter = N - (N & (step - 1));
  for (; l < end_iter;
       l += step) {
    auto key_i = reinterpret_cast<vec_t*>(key[batch_idx][l].data());
    auto value_i = reinterpret_cast<vec_t*>(value[batch_idx][l].data());

    compute_loop<scalar_t, vec_t, kBlockSizeK, kBlockSizeQ, BUFFER_SIZE>(query_block, key_i, value_i, m_prime, s_prime, buffer, K);
  }

  if (l < N) {
    // TODO: unroll this in a generic manner
    l = N - (N & (step - 1)) + threadIdx.x * (kBlockSizeK / 2);
    end_iter = N - (N & (step / 2 - 1));
    for (; l < end_iter;
         l += step / 2) {
      auto key_i = reinterpret_cast<vec_t*>(key[batch_idx][l].data());
      auto value_i = reinterpret_cast<vec_t*>(value[batch_idx][l].data());
      compute_loop<scalar_t, vec_t, kBlockSizeK / 2, kBlockSizeQ, BUFFER_SIZE>(query_block, key_i, value_i, m_prime, s_prime, buffer, K);
    }

    l = N - (N & (step / 2 - 1)) + threadIdx.x * (kBlockSizeK / 4);
    end_iter = N - (N & (step / 4 - 1));
    for (; l < end_iter;
         l += step / 4) {
      auto key_i = reinterpret_cast<vec_t*>(key[batch_idx][l].data());
      auto value_i = reinterpret_cast<vec_t*>(value[batch_idx][l].data());
      compute_loop<scalar_t, vec_t, kBlockSizeK / 4, kBlockSizeQ, BUFFER_SIZE>(query_block, key_i, value_i, m_prime, s_prime, buffer, K);
    }

    l = N - (N & (step / 4 - 1)) + threadIdx.x * (kBlockSizeK / 8);
    end_iter = N - (N & (step / 8 - 1));
    for (; l < end_iter;
         l += step / 8) {
      auto key_i = reinterpret_cast<vec_t*>(key[batch_idx][l].data());
      auto value_i = reinterpret_cast<vec_t*>(value[batch_idx][l].data());
      compute_loop<scalar_t, vec_t, kBlockSizeK / 8, kBlockSizeQ, BUFFER_SIZE>(query_block, key_i, value_i, m_prime, s_prime, buffer, K);
    }

    l = N - (N & (step / 8 - 1)) + threadIdx.x * (kBlockSizeK / 16);
    end_iter = N - (N & (step / 16 - 1));
    for (; l < end_iter;
         l += step / 16) {
      auto key_i = reinterpret_cast<vec_t*>(key[batch_idx][l].data());
      auto value_i = reinterpret_cast<vec_t*>(value[batch_idx][l].data());
      compute_loop<scalar_t, vec_t, kBlockSizeK / 16, kBlockSizeQ, BUFFER_SIZE>(query_block, key_i, value_i, m_prime, s_prime, buffer, K);
    }

    l = N - (N & (step / 16 - 1)) + threadIdx.x;
    for (; l < N;
         l += blockDim.x) {
      auto key_i = reinterpret_cast<vec_t*>(key[batch_idx][l].data());
      auto value_i = reinterpret_cast<vec_t*>(value[batch_idx][l].data());
      compute_loop<scalar_t, vec_t, 1, kBlockSizeQ, BUFFER_SIZE>(query_block, key_i, value_i, m_prime, s_prime, buffer, K);
    }
  }

  aggregate_coeffs<scalar_t, vec_t, kBlockSizeQ, WARP_SIZE, BUFFER_SIZE>(m_prime, s_prime, buffer, K);

  for (int64_t k = threadIdx.x; k < K / kVecSize; k += blockDim.x) {
    vec_t tmp;

#pragma unroll
    for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
      tmp = buffer[q_item_idx][k];
      iDiv<scalar_t>(s_prime[q_item_idx], &tmp);

      output_block[q_item_idx][k] = tmp;
    }
  }
}

at::Tensor attention(
    const at::Tensor& query,
    const at::Tensor& key,
    const at::Tensor& value
    // const at::Tensor& mask
) {
  TORCH_CHECK(query.dim() == key.dim());
  // TORCH_CHECK(query.dim() == mask.dim());
  TORCH_CHECK(query.dim() == 3);
  TORCH_CHECK(query.size(2) == key.size(2));
  TORCH_CHECK(query.size(0) == key.size(0));
  // TORCH_CHECK(query.size(1) == mask.size(1));
  // TORCH_CHECK(query.size(2) == mask.size(2));
  // TORCH_CHECK(query.size(0) == mask.size(0));

  /*
  TORCH_CHECK(!a.is_cuda(), "a must be a CPU tensor");
  TORCH_CHECK(!b.is_cuda(), "b must be a CPU tensor");
  TORCH_CHECK(!mask.is_cuda(), "mask must be a CPU tensor");

  TORCH_CHECK(!a.is_sparse(), "a must be a dense tensor");
  TORCH_CHECK(!b.is_sparse(), "b must be a dense tensor");
  //TORCH_CHECK(mask.is_sparse(), "mask must be a sparse tensor");
  */
  TORCH_CHECK(query.is_contiguous());
  TORCH_CHECK(key.is_contiguous());
  TORCH_CHECK(value.is_contiguous());

  at::cuda::CUDAGuard device_guard(query.device());

  int64_t B = query.size(0);
  int64_t M = query.size(1);
  int64_t N = key.size(1);
  int64_t K = query.size(2);

  at::Tensor res = at::zeros({B, M, K}, query.options());

  constexpr int WARP_SIZE = 4;

  constexpr int kBlockSizeK = 32;
  constexpr int kBlockSizeQ = 2;

  constexpr int TILE_SIZE = 32;
  constexpr int BUFFER_SIZE = 8;

  dim3 grid(ceil_div(M, int64_t(TILE_SIZE)), B);
  dim3 block(WARP_SIZE, TILE_SIZE / kBlockSizeQ);
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  using scalar_t = float;
  // AT_DISPATCH_FLOATING_TYPES(
  // query.scalar_type(), "attention_kernel", [&] {

  if ((K % 4) == 0) {
    attention_kernel<scalar_t, float4, kBlockSizeK, kBlockSizeQ, WARP_SIZE, BUFFER_SIZE>
        <<<grid, block, 0, stream>>>(
            res.packed_accessor<scalar_t, 3>(),
            query.packed_accessor<scalar_t, 3>(),
            key.packed_accessor<scalar_t, 3>(),
            value.packed_accessor<scalar_t, 3>()
            // buffer.accessor<scalar_t, 3>()
            // idxs.accessor<int64_t, 2>()
        );
  } else if ((K % 2) == 0) {
    attention_kernel<scalar_t, float2, kBlockSizeK, kBlockSizeQ, WARP_SIZE, BUFFER_SIZE>
        <<<grid, block, 0, stream>>>(
            res.packed_accessor<scalar_t, 3>(),
            query.packed_accessor<scalar_t, 3>(),
            key.packed_accessor<scalar_t, 3>(),
            value.packed_accessor<scalar_t, 3>()
            // buffer.accessor<scalar_t, 3>()
            // idxs.accessor<int64_t, 2>()
        );

  } else {
    attention_kernel<scalar_t, float, kBlockSizeK, kBlockSizeQ, WARP_SIZE, BUFFER_SIZE>
        <<<grid, block, 0, stream>>>(
            res.packed_accessor<scalar_t, 3>(),
            query.packed_accessor<scalar_t, 3>(),
            key.packed_accessor<scalar_t, 3>(),
            value.packed_accessor<scalar_t, 3>()
            // buffer.accessor<scalar_t, 3>()
            // idxs.accessor<int64_t, 2>()
        );

  }
  //});

  return res;
}

} // namespace

TORCH_LIBRARY_IMPL(xformers, CUDA, m) {
  m.impl(
      TORCH_SELECTIVE_NAME("xformers::efficient_attention"),
      TORCH_FN(attention));
}
