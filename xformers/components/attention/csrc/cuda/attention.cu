#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <torch/library.h>
#include <cmath>
#include <vector>

#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>
#include <ATen/cuda/Atomic.cuh>

#include "sputnik/vector_utils.h"

namespace {

template <typename integer>
constexpr __host__ __device__ inline integer ceil_div(integer n, integer m) {
  return (n + m - 1) / m;
}

template <typename scalar_t>
constexpr __host__ __device__ bool integerIsPowerOf2(scalar_t v) {
  return (v && !(v & (v - 1)));
}

template <typename scalar_t>
__device__ __forceinline__ void iMul(scalar_t x1, float4* out) {
  out[0].x *= x1;
  out[0].y *= x1;
  out[0].z *= x1;
  out[0].w *= x1;
}

template <typename scalar_t>
__device__ __forceinline__ void iMul(scalar_t x1, float2* out) {
  out[0].x *= x1;
  out[0].y *= x1;
}

template <typename scalar_t>
__device__ __forceinline__ void iMul(scalar_t x1, float* out) {
  out[0] *= x1;
}

template <typename scalar_t>
__device__ __forceinline__ void iDiv(scalar_t x1, float4* out) {
  out[0].x /= x1;
  out[0].y /= x1;
  out[0].z /= x1;
  out[0].w /= x1;
}

template <typename scalar_t>
__device__ __forceinline__ void iDiv(scalar_t x1, float2* out) {
  out[0].x /= x1;
  out[0].y /= x1;
}

template <typename scalar_t>
__device__ __forceinline__ void iDiv(scalar_t x1, float* out) {
  out[0] /= x1;
}

template <typename scalar_t>
__device__ __forceinline__ void myGpuAtomicAdd(scalar_t* address, float4 val) {
  gpuAtomicAdd(address + 0, val.x);
  gpuAtomicAdd(address + 1, val.y);
  gpuAtomicAdd(address + 2, val.z);
  gpuAtomicAdd(address + 3, val.w);
}

template <typename scalar_t>
__device__ __forceinline__ void myGpuAtomicAdd(scalar_t* address, float2 val) {
  gpuAtomicAdd(address + 0, val.x);
  gpuAtomicAdd(address + 1, val.y);
}

template <typename scalar_t>
__device__ __forceinline__ void myGpuAtomicAdd(scalar_t* address, float val) {
  gpuAtomicAdd(address, val);
}

template <typename scalar_t, int WARP_SIZE>
__device__ __forceinline__ scalar_t warpSum(scalar_t val) {
  for (int stride = WARP_SIZE / 2; stride > 0; stride >>= 1) {
    val += __shfl_xor_sync(0xffffffff, val, stride, WARP_SIZE);
  }
  return val;
}

template <typename scalar_t, int WARP_SIZE>
__device__ __forceinline__ float2 warpSum(float2 val) {
  for (int stride = WARP_SIZE / 2; stride > 0; stride >>= 1) {
    val.x += __shfl_xor_sync(0xffffffff, val.x, stride, WARP_SIZE);
    val.y += __shfl_xor_sync(0xffffffff, val.y, stride, WARP_SIZE);
  }
  return val;
}

template <typename scalar_t, int WARP_SIZE>
__device__ __forceinline__ float4 warpSum(float4 val) {
  for (int stride = WARP_SIZE / 2; stride > 0; stride >>= 1) {
    val.x += __shfl_xor_sync(0xffffffff, val.x, stride, WARP_SIZE);
    val.y += __shfl_xor_sync(0xffffffff, val.y, stride, WARP_SIZE);
    val.z += __shfl_xor_sync(0xffffffff, val.z, stride, WARP_SIZE);
    val.w += __shfl_xor_sync(0xffffffff, val.w, stride, WARP_SIZE);
  }
  return val;
}

template <typename scalar_t, int WARP_SIZE>
__device__ __forceinline__ scalar_t warpMax(scalar_t val) {
  for (int stride = WARP_SIZE / 2; stride > 0; stride >>= 1) {
    scalar_t tmp = __shfl_xor_sync(0xffffffff, val, stride, WARP_SIZE);
    val = tmp > val ? tmp : val;
  }
  return val;
}

template <typename scalar_t, typename vec_t, int kBlockSizeK, int kBlockSizeQ>
__device__ void compute_dot(
    vec_t* queries[kBlockSizeQ],
    vec_t* keys,
    scalar_t out[kBlockSizeQ][kBlockSizeK],
    int64_t K) {
  constexpr int kVecSize = sizeof(vec_t) / sizeof(scalar_t);
  scalar_t scale = 1.0; // / std::sqrt(scalar_t(K));
  vec_t q_i[kBlockSizeQ];
  for (int64_t k = 0; k < K / kVecSize; k += 1) {
#pragma unroll
    for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
      q_i[q_item_idx] = __ldg(queries[q_item_idx] + k);
      iMul(scale, q_i + q_item_idx);
    }
#pragma unroll
    for (int64_t k_item_idx = 0; k_item_idx < kBlockSizeK; k_item_idx++) {
      vec_t k_i = keys[k + K / kVecSize * k_item_idx];
#pragma unroll
      for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
        sputnik::VectorCompute<vec_t>::Dot(
            q_i[q_item_idx], k_i, &out[q_item_idx][k_item_idx]);
      }
    }
  }
}

template <
    typename scalar_t,
    typename vec_t,
    int kBlockSizeK,
    int kBlockSizeQ,
    int BUFFER_SIZE>
__device__ void compute_final_mult(
    vec_t* vi,
    scalar_t s_delta[kBlockSizeQ][kBlockSizeK],
    scalar_t m_delta[kBlockSizeQ],
    vec_t buffer[kBlockSizeQ][BUFFER_SIZE] /*TODO [BUFFER_SIZE limitation]*/,
    int64_t K) {
  constexpr int kVecSize = sizeof(vec_t) / sizeof(scalar_t);

  for (int64_t k = 0; k < K / kVecSize; k += 1) {
#pragma unroll
    for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
      iMul<scalar_t>(m_delta[q_item_idx], &buffer[q_item_idx][k]);
    }
#pragma unroll
    for (int64_t k_item_idx = 0; k_item_idx < kBlockSizeK; k_item_idx++) {
      vec_t tmp2 = vi[k + K / kVecSize * k_item_idx];

#pragma unroll
      for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
        sputnik::VectorCompute<vec_t>::FMA(
            s_delta[q_item_idx][k_item_idx], tmp2, &buffer[q_item_idx][k]);
      }
    }
  }
}

template <typename scalar_t, int kBlockSizeK, int kBlockSizeQ>
__device__ __forceinline__ void compute_max(
    scalar_t a[kBlockSizeQ][kBlockSizeK],
    scalar_t b[kBlockSizeQ],
    scalar_t out[kBlockSizeQ]) {
#pragma unroll
  for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
    out[q_item_idx] =
        a[q_item_idx][0] > b[q_item_idx] ? a[q_item_idx][0] : b[q_item_idx];
#pragma unroll
    for (int64_t k_item_idx = 1; k_item_idx < kBlockSizeK; k_item_idx++) {
      out[q_item_idx] = a[q_item_idx][k_item_idx] > out[q_item_idx]
          ? a[q_item_idx][k_item_idx]
          : out[q_item_idx];
    }
  }
}

template <typename scalar_t, int kBlockSizeK, int kBlockSizeQ>
__device__ __forceinline__ void compute_scaling_coeffs(
    scalar_t m_i[kBlockSizeQ],
    scalar_t m_prime[kBlockSizeQ],
    scalar_t si[kBlockSizeQ][kBlockSizeK],
    scalar_t m_delta[kBlockSizeQ],
    scalar_t s_delta[kBlockSizeQ][kBlockSizeK]) {
#pragma unroll
  for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++)
    m_delta[q_item_idx] = std::exp(m_prime[q_item_idx] - m_i[q_item_idx]);
#pragma unroll
  for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++)
#pragma unroll
    for (int64_t k_item_idx = 0; k_item_idx < kBlockSizeK; k_item_idx++)
      s_delta[q_item_idx][k_item_idx] =
          std::exp(si[q_item_idx][k_item_idx] - m_i[q_item_idx]);
}

template <typename scalar_t, int kBlockSizeK, int kBlockSizeQ>
__device__ __forceinline__ void update_scaling_coeffs(
    scalar_t m_delta[kBlockSizeQ],
    scalar_t m_i[kBlockSizeQ],
    scalar_t s_delta[kBlockSizeQ][kBlockSizeK],
    scalar_t m_prime[kBlockSizeQ],
    scalar_t s_prime[kBlockSizeQ]) {
#pragma unroll
  for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
    s_prime[q_item_idx] = s_prime[q_item_idx] * m_delta[q_item_idx];
#pragma unroll
    for (int64_t k_item_idx = 0; k_item_idx < kBlockSizeK; k_item_idx++)
      s_prime[q_item_idx] += s_delta[q_item_idx][k_item_idx];

    m_prime[q_item_idx] = m_i[q_item_idx];
  }
}

template <
    typename scalar_t,
    typename vec_t,
    int kBlockSizeK,
    int kBlockSizeQ,
    int BUFFER_SIZE>
__device__ void compute_loop(
    vec_t* query_block[kBlockSizeQ],
    vec_t* key_i,
    vec_t* value_i,
    scalar_t m_prime[kBlockSizeQ],
    scalar_t s_prime[kBlockSizeQ],
    vec_t buffer[kBlockSizeQ][BUFFER_SIZE] /*TODO [BUFFER_SIZE limitation]*/,
    int64_t K) {
  scalar_t si[kBlockSizeQ][kBlockSizeK] = {0};
  compute_dot<scalar_t, vec_t, kBlockSizeK, kBlockSizeQ>(
      query_block, key_i, si, K);

  scalar_t m_i[kBlockSizeQ];
  compute_max<scalar_t, kBlockSizeK, kBlockSizeQ>(si, m_prime, m_i);

  scalar_t m_delta[kBlockSizeQ];
  scalar_t s_delta[kBlockSizeQ][kBlockSizeK];

  compute_scaling_coeffs<scalar_t, kBlockSizeK, kBlockSizeQ>(
      m_i, m_prime, si, m_delta, s_delta);

  compute_final_mult<scalar_t, vec_t, kBlockSizeK, kBlockSizeQ, BUFFER_SIZE>(
      value_i, s_delta, m_delta, buffer, K);

  update_scaling_coeffs<scalar_t, kBlockSizeK, kBlockSizeQ>(
      m_delta, m_i, s_delta, m_prime, s_prime);
}

template <
    typename scalar_t,
    typename vec_t,
    int kBlockSizeQ,
    int WARP_SIZE,
    int BUFFER_SIZE>
__device__ __forceinline__ void aggregate_coeffs(
    scalar_t m_prime[kBlockSizeQ],
    scalar_t s_prime[kBlockSizeQ],
    vec_t buffer[kBlockSizeQ][BUFFER_SIZE] /*TODO [BUFFER_SIZE limitation]*/,
    int64_t K) {
  constexpr int kVecSize = sizeof(vec_t) / sizeof(scalar_t);
  for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
    scalar_t m_i = m_prime[q_item_idx];
    scalar_t s_i = s_prime[q_item_idx];
    m_prime[q_item_idx] = warpMax<scalar_t, WARP_SIZE>(m_prime[q_item_idx]);
    scalar_t m_delta = std::exp(m_i - m_prime[q_item_idx]);
    scalar_t s_delta = s_i * m_delta;
    s_delta = warpSum<scalar_t, WARP_SIZE>(s_delta);
    s_prime[q_item_idx] = s_delta;
    for (int64_t k = 0; k < K / kVecSize; k += 1) {
      vec_t tmp = buffer[q_item_idx][k];
      iMul<scalar_t>(m_delta, &tmp);
      tmp = warpSum<vec_t, WARP_SIZE>(tmp);
      buffer[q_item_idx][k] = tmp;
    }
  }
}

template <
    bool first,
    typename scalar_t,
    typename vec_t,
    int kBlockSizeK,
    int kBlockSizeQ,
    int BUFFER_SIZE,
    int WARP_SIZE>
struct UnrollLoop {
  static __device__ __forceinline__ void eval(
      vec_t* query_block[kBlockSizeQ],
      at::TensorAccessor<scalar_t, 2> key,
      at::TensorAccessor<scalar_t, 2> value,
      scalar_t m_prime[kBlockSizeQ],
      scalar_t s_prime[kBlockSizeQ],
      vec_t buffer[kBlockSizeQ][BUFFER_SIZE] /*TODO [BUFFER_SIZE limitation]*/,
      int64_t K,
      int64_t N) {
    constexpr int64_t step = kBlockSizeK * WARP_SIZE;
    int64_t l;
    if (first) {
      l = threadIdx.x * kBlockSizeK;
    } else {
      l = N - (N & (2 * step - 1)) + threadIdx.x * kBlockSizeK;
    }
    // this is equivalent to N - N % step, but faster
    // guaranteed to be the same as step is a power of 2
    int64_t end_iter = N - (N & (step - 1));
    // if (l < end_iter) {
    {
      for (; l < end_iter; l += step) {
        auto key_i = reinterpret_cast<vec_t*>(key[l].data());
        auto value_i = reinterpret_cast<vec_t*>(value[l].data());

        compute_loop<scalar_t, vec_t, kBlockSizeK, kBlockSizeQ, BUFFER_SIZE>(
            query_block, key_i, value_i, m_prime, s_prime, buffer, K);
      }
    }
    {
      UnrollLoop<
          false,
          scalar_t,
          vec_t,
          kBlockSizeK / 2,
          kBlockSizeQ,
          BUFFER_SIZE,
          WARP_SIZE>::
          eval(query_block, key, value, m_prime, s_prime, buffer, K, N);
    }
  }
};

template <
    bool first,
    typename scalar_t,
    typename vec_t,
    int kBlockSizeQ,
    int BUFFER_SIZE,
    int WARP_SIZE>
struct UnrollLoop<
    first,
    scalar_t,
    vec_t,
    0,
    kBlockSizeQ,
    BUFFER_SIZE,
    WARP_SIZE> {
  static __device__ __forceinline__ void eval(
      vec_t* query_block[kBlockSizeQ],
      at::TensorAccessor<scalar_t, 2> key,
      at::TensorAccessor<scalar_t, 2> value,
      scalar_t m_prime[kBlockSizeQ],
      scalar_t s_prime[kBlockSizeQ],
      vec_t buffer[kBlockSizeQ][BUFFER_SIZE] /*TODO [BUFFER_SIZE limitation]*/,
      int64_t K,
      int64_t N) {}
};

template <
    typename scalar_t,
    typename vec_t,
    int kBlockSizeK,
    int kBlockSizeQ,
    int WARP_SIZE,
    int BUFFER_SIZE>
__global__ void attention_kernel(
    at::PackedTensorAccessor<scalar_t, 3> output,
    at::PackedTensorAccessor<scalar_t, 3> query,
    at::PackedTensorAccessor<scalar_t, 3> key,
    at::PackedTensorAccessor<scalar_t, 3> value) {
  constexpr int kVecSize = sizeof(vec_t) / sizeof(scalar_t);
  static_assert(
      integerIsPowerOf2(kBlockSizeK * WARP_SIZE),
      "kBlockSizeK * WARP_SIZE should be a power of 2");
  int64_t K = query.size(2);
  int64_t B = query.size(0);
  int64_t M = query.size(1);
  int64_t N = key.size(1);

  int64_t batch_idx = blockIdx.y;
  int64_t query_idx =
      blockIdx.x * (blockDim.y * kBlockSizeQ) + threadIdx.y * kBlockSizeQ;

  if (query_idx >= M)
    return;

  vec_t* query_block[kBlockSizeQ];
  vec_t* output_block[kBlockSizeQ];
  // TODO [BUFFER_SIZE limitation]: the current strategy assumes a
  // statically-known size for K. Ideally we would like to remove this
  // limitation in the future, so that any K is supported
  vec_t buffer[kBlockSizeQ][BUFFER_SIZE] = {0};
  scalar_t s_prime[kBlockSizeQ] = {0};
  scalar_t m_prime[kBlockSizeQ];
  for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
    int64_t index = query_idx + q_item_idx;
    index = index >= M ? M - 1 : index;
    query_block[q_item_idx] =
        reinterpret_cast<vec_t*>(query[batch_idx][index].data());
    output_block[q_item_idx] =
        reinterpret_cast<vec_t*>(output[batch_idx][index].data());
    m_prime[q_item_idx] = -std::numeric_limits<scalar_t>::infinity();
  }
#if 0
  // this for now makes things slower
  UnrollLoop<true, scalar_t, vec_t, kBlockSizeK, kBlockSizeQ, BUFFER_SIZE, WARP_SIZE>::eval(query_block, key[batch_idx], value[batch_idx], m_prime, s_prime, buffer, K, N);
#else
  int64_t l = threadIdx.x * kBlockSizeK;
  constexpr int64_t step = kBlockSizeK * WARP_SIZE;
  // this is equivalent to N - N % step, but faster
  // guaranteed to be the same as step is a power of 2
  int64_t end_iter = N - (N & (step - 1));
  for (; l < end_iter; l += step) {
    auto key_i = reinterpret_cast<vec_t*>(key[batch_idx][l].data());
    auto value_i = reinterpret_cast<vec_t*>(value[batch_idx][l].data());

    compute_loop<scalar_t, vec_t, kBlockSizeK, kBlockSizeQ, BUFFER_SIZE>(
        query_block, key_i, value_i, m_prime, s_prime, buffer, K);
  }

  {
    // TODO: unroll this in a generic manner
    l = N - (N & (step - 1)) + threadIdx.x * (kBlockSizeK / 2);
    end_iter = N - (N & (step / 2 - 1));
    for (; l < end_iter; l += step / 2) {
      auto key_i = reinterpret_cast<vec_t*>(key[batch_idx][l].data());
      auto value_i = reinterpret_cast<vec_t*>(value[batch_idx][l].data());
      compute_loop<scalar_t, vec_t, kBlockSizeK / 2, kBlockSizeQ, BUFFER_SIZE>(
          query_block, key_i, value_i, m_prime, s_prime, buffer, K);
    }

    l = N - (N & (step / 2 - 1)) + threadIdx.x * (kBlockSizeK / 4);
    end_iter = N - (N & (step / 4 - 1));
    for (; l < end_iter; l += step / 4) {
      auto key_i = reinterpret_cast<vec_t*>(key[batch_idx][l].data());
      auto value_i = reinterpret_cast<vec_t*>(value[batch_idx][l].data());
      compute_loop<scalar_t, vec_t, kBlockSizeK / 4, kBlockSizeQ, BUFFER_SIZE>(
          query_block, key_i, value_i, m_prime, s_prime, buffer, K);
    }

    l = N - (N & (step / 4 - 1)) + threadIdx.x * (kBlockSizeK / 8);
    end_iter = N - (N & (step / 8 - 1));
    for (; l < end_iter; l += step / 8) {
      auto key_i = reinterpret_cast<vec_t*>(key[batch_idx][l].data());
      auto value_i = reinterpret_cast<vec_t*>(value[batch_idx][l].data());
      compute_loop<scalar_t, vec_t, kBlockSizeK / 8, kBlockSizeQ, BUFFER_SIZE>(
          query_block, key_i, value_i, m_prime, s_prime, buffer, K);
    }

    l = N - (N & (step / 8 - 1)) + threadIdx.x * (kBlockSizeK / 16);
    end_iter = N - (N & (step / 16 - 1));
    for (; l < end_iter; l += step / 16) {
      auto key_i = reinterpret_cast<vec_t*>(key[batch_idx][l].data());
      auto value_i = reinterpret_cast<vec_t*>(value[batch_idx][l].data());
      compute_loop<scalar_t, vec_t, kBlockSizeK / 16, kBlockSizeQ, BUFFER_SIZE>(
          query_block, key_i, value_i, m_prime, s_prime, buffer, K);
    }

    l = N - (N & (step / 16 - 1)) + threadIdx.x;
    for (; l < N; l += blockDim.x) {
      auto key_i = reinterpret_cast<vec_t*>(key[batch_idx][l].data());
      auto value_i = reinterpret_cast<vec_t*>(value[batch_idx][l].data());
      compute_loop<scalar_t, vec_t, 1, kBlockSizeQ, BUFFER_SIZE>(
          query_block, key_i, value_i, m_prime, s_prime, buffer, K);
    }
  }
#endif

  aggregate_coeffs<scalar_t, vec_t, kBlockSizeQ, WARP_SIZE, BUFFER_SIZE>(
      m_prime, s_prime, buffer, K);

  for (int64_t k = threadIdx.x; k < K / kVecSize; k += blockDim.x) {
    vec_t tmp;

#pragma unroll
    for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
      tmp = buffer[q_item_idx][k];
      iDiv<scalar_t>(s_prime[q_item_idx], &tmp);

      output_block[q_item_idx][k] = tmp;
    }
  }
}

at::Tensor attention(
    const at::Tensor& query,
    const at::Tensor& key,
    const at::Tensor& value
    // const at::Tensor& mask
) {
  TORCH_CHECK(query.dim() == key.dim());
  TORCH_CHECK(query.dim() == value.dim());
  // TORCH_CHECK(query.dim() == mask.dim());
  TORCH_CHECK(query.dim() == 3);
  TORCH_CHECK(query.size(2) == key.size(2));
  TORCH_CHECK(query.size(0) == key.size(0));

  TORCH_CHECK(query.size(0) == value.size(0));
  TORCH_CHECK(key.size(1) == value.size(1));
  TORCH_CHECK(
      query.size(2) ==
      value.size(2)); // TODO: drop this limitation in the future

  TORCH_CHECK(query.is_cuda(), "query must be a CUDA tensor");
  TORCH_CHECK(key.is_cuda(), "key must be a CUDA tensor");
  TORCH_CHECK(value.is_cuda(), "value must be a CUDA tensor");

  TORCH_CHECK(!query.is_sparse(), "query must be a dense tensor");
  TORCH_CHECK(!key.is_sparse(), "key must be a dense tensor");
  TORCH_CHECK(!value.is_sparse(), "value must be a dense tensor");

  // TODO drop this limitation in the future
  TORCH_CHECK(query.is_contiguous());
  TORCH_CHECK(key.is_contiguous());
  TORCH_CHECK(value.is_contiguous());

  // TODO: support other dtypes in the future
  TORCH_CHECK(
      query.scalar_type() == at::ScalarType::Float,
      "Only float32 type is supported for now");

  at::cuda::CUDAGuard device_guard(query.device());

  int64_t B = query.size(0);
  int64_t M = query.size(1);
  int64_t N = key.size(1);
  int64_t K = query.size(2);

  at::Tensor res = at::zeros({B, M, K}, query.options());

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  constexpr int WARP_SIZE = 4;

  constexpr int kBlockSizeK = 32;
  constexpr int kBlockSizeQ = 2;

  constexpr int TILE_SIZE = 32;
  constexpr int BUFFER_SIZE = 8;

  dim3 grid(ceil_div(M, int64_t(TILE_SIZE)), B);
  dim3 block(WARP_SIZE, TILE_SIZE / kBlockSizeQ);

  using scalar_t = float;

  if ((K % 4) == 0) {
    TORCH_CHECK(
        K / 4 <= BUFFER_SIZE,
        "For now only a certain number of K values are supported. Let us know if you hit this and we will fix it");
    attention_kernel<
        scalar_t,
        float4,
        kBlockSizeK,
        kBlockSizeQ,
        WARP_SIZE,
        BUFFER_SIZE><<<grid, block, 0, stream>>>(
        res.packed_accessor<scalar_t, 3>(),
        query.packed_accessor<scalar_t, 3>(),
        key.packed_accessor<scalar_t, 3>(),
        value.packed_accessor<scalar_t, 3>());
  } else if ((K % 2) == 0) {
    TORCH_CHECK(
        K / 2 <= BUFFER_SIZE,
        "For now only a certain number of K values are supported. Let us know if you hit this and we will fix it");
    attention_kernel<
        scalar_t,
        float2,
        kBlockSizeK,
        kBlockSizeQ,
        WARP_SIZE,
        BUFFER_SIZE><<<grid, block, 0, stream>>>(
        res.packed_accessor<scalar_t, 3>(),
        query.packed_accessor<scalar_t, 3>(),
        key.packed_accessor<scalar_t, 3>(),
        value.packed_accessor<scalar_t, 3>());

  } else {
    TORCH_CHECK(
        K <= BUFFER_SIZE,
        "For now only a certain number of K values are supported. Let us know if you hit this and we will fix it");
    attention_kernel<
        scalar_t,
        float,
        kBlockSizeK,
        kBlockSizeQ,
        WARP_SIZE,
        BUFFER_SIZE><<<grid, block, 0, stream>>>(
        res.packed_accessor<scalar_t, 3>(),
        query.packed_accessor<scalar_t, 3>(),
        key.packed_accessor<scalar_t, 3>(),
        value.packed_accessor<scalar_t, 3>());
  }
  AT_CUDA_CHECK(hipGetLastError());

  return res;
}

template <
    typename scalar_t,
    typename vec_t,
    int kBlockSizeQ,
    int kBlockSizeK,
    int BUFFER_SIZE>
__global__ void attention_backward_kernel(
    at::PackedTensorAccessor<scalar_t, 3> grad_q,
    at::PackedTensorAccessor<scalar_t, 3> grad_k,
    at::PackedTensorAccessor<scalar_t, 3> grad_v,
    at::PackedTensorAccessor<scalar_t, 3> grad_out,
    at::PackedTensorAccessor<scalar_t, 3> query,
    at::PackedTensorAccessor<scalar_t, 3> key,
    at::PackedTensorAccessor<scalar_t, 3> value,
    at::PackedTensorAccessor<scalar_t, 2> tmp_sum_i,
    at::PackedTensorAccessor<scalar_t, 2> logsumexp_normalizer) {
  int64_t K = query.size(2);
  int64_t B = query.size(0);
  int64_t M = query.size(1);
  int64_t N = key.size(1);

  constexpr int kVecSize = sizeof(vec_t) / sizeof(scalar_t);

  int64_t batch_idx = blockIdx.y;
  int64_t query_idx =
      blockIdx.x * blockDim.y * kBlockSizeQ + threadIdx.y * kBlockSizeQ;

  if (query_idx >= M)
    return;

  //vec_t temp_buffer[kBlockSizeQ][BUFFER_SIZE] = {0};
  //vec_t temp_grad_q[kBlockSizeQ][BUFFER_SIZE] = {0};

  vec_t* query_block[kBlockSizeQ];
  vec_t* grad_out_block[kBlockSizeQ];
  //vec_t* grad_q_block[kBlockSizeQ];
  scalar_t normalizer[kBlockSizeQ];

  //__shared__ vec_t query_cache[kBlockSizeQ][BUFFER_SIZE];

  for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
    int64_t index = query_idx + q_item_idx;
    index = index >= M ? M - 1 : index;
    query_block[q_item_idx] =
        reinterpret_cast<vec_t*>(query[batch_idx][index].data());
    grad_out_block[q_item_idx] =
        reinterpret_cast<vec_t*>(grad_out[batch_idx][index].data());
    //grad_q_block[q_item_idx] =
    //    reinterpret_cast<vec_t*>(grad_q[batch_idx][index].data());
    normalizer[q_item_idx] = logsumexp_normalizer[batch_idx][index];
    //for (int64_t k = threadIdx.x; k < K / kVecSize; k += blockDim.x) {
    //  query_cache[q_item_idx][k] = query_block[q_item_idx][k];
   // }
  }
  //__syncthreads();

  scalar_t tmp_sum[kBlockSizeQ] = {0};
  for (int64_t l = threadIdx.x * kBlockSizeK; l < N;
       l += blockDim.x * kBlockSizeK) {
    auto key_j = reinterpret_cast<vec_t*>(key[batch_idx][l].data());

    scalar_t attn_v[kBlockSizeQ][kBlockSizeK] = {0};
    compute_dot<scalar_t, vec_t, kBlockSizeK, kBlockSizeQ>(
        //query_cache, key_j, attn_v, K);
        query_block, key_j, attn_v, K);

#pragma unroll
    for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
#pragma unroll
      for (int64_t k_item_idx = 0; k_item_idx < kBlockSizeK; k_item_idx++) {
        attn_v[q_item_idx][k_item_idx] =
            std::exp(attn_v[q_item_idx][k_item_idx] - normalizer[q_item_idx]);
      }
    }

    // now compute grad_q and grad_k
    // first compute the gradient for the self-attention
    // after softmax
    // scalar_t grad_attn_v = 0;
    scalar_t grad_attn_v[kBlockSizeQ][kBlockSizeK] = {0};
    auto value_j = reinterpret_cast<vec_t*>(value[batch_idx][l].data());

    for (int64_t k = 0; k < K / kVecSize; k++) {
      vec_t temp_i[kBlockSizeQ];
#pragma unroll
      for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
        temp_i[q_item_idx] = __ldg(grad_out_block[q_item_idx] + k);
      }

#pragma unroll
      for (int64_t k_item_idx = 0; k_item_idx < kBlockSizeK; k_item_idx++) {
        vec_t v = value_j[k + K / kVecSize * k_item_idx];
        vec_t tt = {0};
#pragma unroll
        for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
          sputnik::VectorCompute<vec_t>::Dot(
              temp_i[q_item_idx], v, &grad_attn_v[q_item_idx][k_item_idx]);
          sputnik::VectorCompute<vec_t>::FMA(attn_v[q_item_idx][k_item_idx], temp_i[q_item_idx], &tt);
        }
        myGpuAtomicAdd(&grad_v[batch_idx][l + k_item_idx][k * kVecSize], tt);
      }
    }

    // those are temporaries for the gradient of the softmax
    //scalar_t tmp[kBlockSizeQ][kBlockSizeK];
#pragma unroll
    for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
#pragma unroll
      for (int64_t k_item_idx = 0; k_item_idx < kBlockSizeK; k_item_idx++) {
        //tmp[q_item_idx][k_item_idx] = attn_v[q_item_idx][k_item_idx] *
        //    grad_attn_v[q_item_idx][k_item_idx];
        //tmp_sum[q_item_idx] += tmp[q_item_idx][k_item_idx];
        tmp_sum[q_item_idx] += attn_v[q_item_idx][k_item_idx] * grad_attn_v[q_item_idx][k_item_idx];
      }
    }
/*
    // grad_q is easy
    for (int64_t k = 0; k < K / kVecSize; k++) {
#pragma unroll
      for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
#pragma unroll
        for (int64_t k_item_idx = 0; k_item_idx < kBlockSizeK; k_item_idx++) {
          vec_t ttt = key_j[k + K / kVecSize * k_item_idx];
          sputnik::VectorCompute<vec_t>::FMA(tmp[q_item_idx][k_item_idx], ttt, &temp_grad_q[q_item_idx][k]);
          sputnik::VectorCompute<vec_t>::FMA(
              attn_v[q_item_idx][k_item_idx], ttt, &temp_buffer[q_item_idx][k]);
        }
      }
    }
*/
    //  but grad_k is a bit trickier
/*
    for (int64_t k = 0; k < K / kVecSize; k++) {
#pragma unroll
      for (int64_t k_item_idx = 0; k_item_idx < kBlockSizeK; k_item_idx++) {
        vec_t res = {0};
#pragma unroll
        for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
          // res += tmp[q_item_idx][k_item_idx] * query_block[q_item_idx][k];
          vec_t qqq = __ldg(query_block[q_item_idx] + k);
          //vec_t qqq = query_cache[q_item_idx][k];
          sputnik::VectorCompute<vec_t>::FMA(tmp[q_item_idx][k_item_idx], qqq, &res);
        }
        myGpuAtomicAdd(&grad_k[batch_idx][l + k_item_idx][k * kVecSize], res);
      }
    }*/
  }
#pragma unroll
  for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
    tmp_sum[q_item_idx] = warpSum<scalar_t, 32>(tmp_sum[q_item_idx]);
    tmp_sum_i[batch_idx][query_idx + q_item_idx] = tmp_sum[q_item_idx];
  }
/*
  for (int64_t l = threadIdx.x * kBlockSizeK; l < N;
       l += blockDim.x * kBlockSizeK) {
    auto key_j = reinterpret_cast<vec_t*>(key[batch_idx][l].data());
    scalar_t attn_v[kBlockSizeQ][kBlockSizeK] = {0};
    compute_dot<scalar_t, vec_t, kBlockSizeK, kBlockSizeQ>(
        query_block, key_j, attn_v, K);

#pragma unroll
    for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
#pragma unroll
      for (int64_t k_item_idx = 0; k_item_idx < kBlockSizeK; k_item_idx++) {
        attn_v[q_item_idx][k_item_idx] =
            std::exp(attn_v[q_item_idx][k_item_idx] - normalizer[q_item_idx]);
      }
    }

    scalar_t grad_attn_v[kBlockSizeQ][kBlockSizeK] = {0};
    auto value_j = reinterpret_cast<vec_t*>(value[batch_idx][l].data());

    for (int64_t k = 0; k < K / kVecSize; k++) {
      vec_t temp_i[kBlockSizeQ];
#pragma unroll
      for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
        temp_i[q_item_idx] = __ldg(grad_out_block[q_item_idx] + k);
      }

#pragma unroll
      for (int64_t k_item_idx = 0; k_item_idx < kBlockSizeK; k_item_idx++) {
        vec_t v = value_j[k + K / kVecSize * k_item_idx];
        vec_t tt = {0};
#pragma unroll
        for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
          sputnik::VectorCompute<vec_t>::Dot(
              temp_i[q_item_idx], v, &grad_attn_v[q_item_idx][k_item_idx]);
        }
      }
    }

    // those are temporaries for the gradient of the softmax
    scalar_t tmp[kBlockSizeQ][kBlockSizeK];
#pragma unroll
    for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
#pragma unroll
      for (int64_t k_item_idx = 0; k_item_idx < kBlockSizeK; k_item_idx++) {
        tmp[q_item_idx][k_item_idx] = attn_v[q_item_idx][k_item_idx] *
            grad_attn_v[q_item_idx][k_item_idx];
      }
    }


    for (int64_t k = 0; k < K / kVecSize; k++) {
#pragma unroll
      for (int64_t k_item_idx = 0; k_item_idx < kBlockSizeK; k_item_idx++) {
        vec_t res = {0};
#pragma unroll
        for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
          //scalar_t ttt = -attn_v[q_item_idx][k_item_idx] * tmp_sum[q_item_idx];
          scalar_t ttt = tmp[q_item_idx][k_item_idx] - attn_v[q_item_idx][k_item_idx] * tmp_sum[q_item_idx];
          vec_t qqq = __ldg(query_block[q_item_idx] + k);
          //vec_t qqq = query_cache[q_item_idx][k];
          sputnik::VectorCompute<vec_t>::FMA(ttt, qqq, &res);
        }
        myGpuAtomicAdd(&grad_k[batch_idx][l + k_item_idx][k * kVecSize], res);
      }
    }
  }*/
  /*
  for (int64_t k = 0; k < K / kVecSize; k++) {
#pragma unroll
    for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
      temp_grad_q[q_item_idx][k] =
          warpSum<scalar_t, 32>(temp_grad_q[q_item_idx][k]);
      temp_buffer[q_item_idx][k] =
          warpSum<scalar_t, 32>(temp_buffer[q_item_idx][k]);
    }
  }
    for (int64_t k = threadIdx.x; k < K / kVecSize; k += blockDim.x) {
#pragma unroll
      for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
        // sputnik::VectorCompute<vec_t>::FMA(-tmp_sum[q_item_idx], temp_buffer[q_item_idx][k], &temp_grad_q[q_item_idx][k]);
        // grad_q_block[q_item_idx][k] = temp_grad_q[q_item_idx][k];
        grad_q_block[q_item_idx][k].x = temp_grad_q[q_item_idx][k].x -
            temp_buffer[q_item_idx][k].x * tmp_sum[q_item_idx];
        grad_q_block[q_item_idx][k].y = temp_grad_q[q_item_idx][k].y -
            temp_buffer[q_item_idx][k].y * tmp_sum[q_item_idx];
        grad_q_block[q_item_idx][k].z = temp_grad_q[q_item_idx][k].z -
            temp_buffer[q_item_idx][k].z * tmp_sum[q_item_idx];
        grad_q_block[q_item_idx][k].w = temp_grad_q[q_item_idx][k].w -
            temp_buffer[q_item_idx][k].w * tmp_sum[q_item_idx];
      }
    }*/
}


template <
    typename scalar_t,
    typename vec_t,
    int kBlockSizeQ,
    int kBlockSizeK,
    int BUFFER_SIZE>
__global__ void attention_backward_kernel3(
    at::PackedTensorAccessor<scalar_t, 3> grad_q,
    at::PackedTensorAccessor<scalar_t, 3> grad_k,
    at::PackedTensorAccessor<scalar_t, 3> grad_v,
    at::PackedTensorAccessor<scalar_t, 3> grad_out,
    at::PackedTensorAccessor<scalar_t, 3> query,
    at::PackedTensorAccessor<scalar_t, 3> key,
    at::PackedTensorAccessor<scalar_t, 3> value,
    at::PackedTensorAccessor<scalar_t, 2> tmp_sum_i,
    at::PackedTensorAccessor<scalar_t, 2> logsumexp_normalizer) {
  int64_t K = query.size(2);
  int64_t B = query.size(0);
  int64_t M = query.size(1);
  int64_t N = key.size(1);

  constexpr int kVecSize = sizeof(vec_t) / sizeof(scalar_t);

  int64_t batch_idx = blockIdx.z;
  int64_t query_idx =
      blockIdx.x * blockDim.x * kBlockSizeQ + threadIdx.x * kBlockSizeQ;

  int64_t l =
      blockIdx.y * blockDim.y * kBlockSizeK + threadIdx.y * kBlockSizeK;

  if (query_idx >= M)
    return;

  if (l >= N)
    return;

  vec_t* query_block[kBlockSizeQ];
  vec_t* grad_out_block[kBlockSizeQ];
  scalar_t normalizer[kBlockSizeQ];
  scalar_t tmp_sum[kBlockSizeQ];

  //__shared__ vec_t query_cache[kBlockSizeQ][BUFFER_SIZE];

  for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
    int64_t index = query_idx + q_item_idx;
    index = index >= M ? M - 1 : index;
    query_block[q_item_idx] =
        reinterpret_cast<vec_t*>(query[batch_idx][index].data());
    grad_out_block[q_item_idx] =
        reinterpret_cast<vec_t*>(grad_out[batch_idx][index].data());
    normalizer[q_item_idx] = logsumexp_normalizer[batch_idx][index];
    tmp_sum[q_item_idx] = tmp_sum_i[batch_idx][index];
    //for (int64_t k = threadIdx.x; k < K / kVecSize; k += blockDim.x) {
    //  query_cache[q_item_idx][k] = query_block[q_item_idx][k];
   // }
  }
  //__syncthreads();

  //scalar_t tmp_sum[kBlockSizeQ] = {0};

  //for (int64_t l = threadIdx.x * kBlockSizeK; l < N;
  //     l += blockDim.x * kBlockSizeK)
  {
    auto key_j = reinterpret_cast<vec_t*>(key[batch_idx][l].data());
    scalar_t attn_v[kBlockSizeQ][kBlockSizeK] = {0};
    //compute_dot<scalar_t, vec_t, kBlockSizeK, kBlockSizeQ>(
    //    query_block, key_j, attn_v, K);

    vec_t q_i[kBlockSizeQ];
    for (int64_t k = 0; k < K / kVecSize; k += 1) {
#pragma unroll
      for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
        q_i[q_item_idx] = __ldg(query_block[q_item_idx] + k);
      }
#pragma unroll
      for (int64_t k_item_idx = 0; k_item_idx < kBlockSizeK; k_item_idx++) {
        vec_t k_i = key_j[k + K / kVecSize * k_item_idx];
#pragma unroll
        for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
          sputnik::VectorCompute<vec_t>::Dot(
              q_i[q_item_idx], k_i, &attn_v[q_item_idx][k_item_idx]);
        }
      }
    }


#pragma unroll
    for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
#pragma unroll
      for (int64_t k_item_idx = 0; k_item_idx < kBlockSizeK; k_item_idx++) {
        attn_v[q_item_idx][k_item_idx] =
            std::exp(attn_v[q_item_idx][k_item_idx] - normalizer[q_item_idx]);
      }
    }

    scalar_t grad_attn_v[kBlockSizeQ][kBlockSizeK] = {0};
    auto value_j = reinterpret_cast<vec_t*>(value[batch_idx][l].data());

    for (int64_t k = 0; k < K / kVecSize; k++) {
      vec_t temp_i[kBlockSizeQ];
#pragma unroll
      for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
        temp_i[q_item_idx] = __ldg(grad_out_block[q_item_idx] + k);
      }

#pragma unroll
      for (int64_t k_item_idx = 0; k_item_idx < kBlockSizeK; k_item_idx++) {
        vec_t v = value_j[k + K / kVecSize * k_item_idx];
#pragma unroll
        for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
          sputnik::VectorCompute<vec_t>::Dot(
              temp_i[q_item_idx], v, &grad_attn_v[q_item_idx][k_item_idx]);
        }
      }
    }

    for (int64_t k = 0; k < K / kVecSize; k++) {
        vec_t res[kBlockSizeQ] = {0};
#pragma unroll
      for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
#pragma unroll
        for (int64_t k_item_idx = 0; k_item_idx < kBlockSizeK; k_item_idx++) {
          vec_t ttt = key_j[k + K / kVecSize * k_item_idx];
          scalar_t ttmp = attn_v[q_item_idx][k_item_idx] * grad_attn_v[q_item_idx][k_item_idx] - attn_v[q_item_idx][k_item_idx] * tmp_sum[q_item_idx];
          sputnik::VectorCompute<vec_t>::FMA(ttmp, ttt, &res[q_item_idx]);
        }
      }
#pragma unroll
      for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
        myGpuAtomicAdd(&grad_q[batch_idx][query_idx + q_item_idx][k * kVecSize], res[q_item_idx]);

      }
    }


    for (int64_t k = 0; k < K / kVecSize; k++) {
#pragma unroll
      for (int64_t k_item_idx = 0; k_item_idx < kBlockSizeK; k_item_idx++) {
        vec_t res = {0};
#pragma unroll
        for (int64_t q_item_idx = 0; q_item_idx < kBlockSizeQ; q_item_idx++) {
          //scalar_t ttt = -attn_v[q_item_idx][k_item_idx] * tmp_sum[q_item_idx];
          scalar_t ttt = attn_v[q_item_idx][k_item_idx] * grad_attn_v[q_item_idx][k_item_idx] - attn_v[q_item_idx][k_item_idx] * tmp_sum[q_item_idx];
          vec_t qqq = __ldg(query_block[q_item_idx] + k);
          //vec_t qqq = query_cache[q_item_idx][k];
          sputnik::VectorCompute<vec_t>::FMA(ttt, qqq, &res);
        }
        myGpuAtomicAdd(&grad_k[batch_idx][l + k_item_idx][k * kVecSize], res);
      }
    }
  }

}


template <
    typename scalar_t,
    typename vec_t,
    int kBlockSizeQ,
    int kBlockSizeK,
    int BUFFER_SIZE>
__global__ void attention_backward_kernel2(
    at::PackedTensorAccessor<scalar_t, 3> grad_q,
    at::PackedTensorAccessor<scalar_t, 3> grad_k,
    at::PackedTensorAccessor<scalar_t, 3> grad_v,
    at::PackedTensorAccessor<scalar_t, 3> grad_out,
    at::PackedTensorAccessor<scalar_t, 3> query,
    at::PackedTensorAccessor<scalar_t, 3> key,
    at::PackedTensorAccessor<scalar_t, 3> value,
    at::PackedTensorAccessor<scalar_t, 2> tmp_sum_i,
    at::PackedTensorAccessor<scalar_t, 2> logsumexp_normalizer) {
  int64_t K = query.size(2);
  int64_t B = query.size(0);
  int64_t M = query.size(1);
  int64_t N = key.size(1);

  constexpr int kVecSize = sizeof(vec_t) / sizeof(scalar_t);

  int64_t batch_idx = blockIdx.z;
  int64_t query_idx =
      blockIdx.x * blockDim.x + threadIdx.x;

  int64_t l =
      blockIdx.y * blockDim.y + threadIdx.y;

  if (query_idx >= M)
    return;

  if (l >= N)
    return;

  scalar_t normalizer;
  scalar_t tmp_sum;

  constexpr int KS1 = 16;
  constexpr int KS2 = 16;

  __shared__ vec_t query_cache[KS1][BUFFER_SIZE];
  __shared__ vec_t key_cache[KS2][BUFFER_SIZE];
  //__shared__ vec_t value_cache[KS2][BUFFER_SIZE];
  //__shared__ vec_t grad_out_cache[KS1][BUFFER_SIZE];
  __shared__ scalar_t fact[KS1][KS2];

  auto qb = reinterpret_cast<vec_t*>(query[batch_idx][query_idx].data());
  auto kb = reinterpret_cast<vec_t*>(key[batch_idx][l].data());
  auto vb = reinterpret_cast<vec_t*>(value[batch_idx][l].data());
  auto gb = reinterpret_cast<vec_t*>(grad_out[batch_idx][query_idx].data());

  //__shared__ vec_t tmp_grad1[KS1][BUFFER_SIZE];
  //__shared__ vec_t tmp_grad2[KS2][BUFFER_SIZE];
  //vec_t query_cache[KS1][BUFFER_SIZE];
  //vec_t key_cache[KS2][BUFFER_SIZE];

  //vec_t tmp_grad1[KS1][BUFFER_SIZE] = {0};
  //vec_t tmp_grad2[KS2][BUFFER_SIZE] = {0};

  vec_t zero = {0};
  for (int64_t k = 0; k < K / kVecSize; k++) {
    query_cache[threadIdx.x][k] = qb[k];
    key_cache[threadIdx.y][k] = kb[k];
    //value_cache[threadIdx.y][k] = vb[k];
    //grad_out_cache[threadIdx.x][k] = gb[k];
    //tmp_grad1[threadIdx.x][k] = zero;
    //tmp_grad2[threadIdx.y][k] = zero;
  }
  //__syncwarp();
  //__syncthreads();


  normalizer = logsumexp_normalizer[batch_idx][query_idx];
  tmp_sum = tmp_sum_i[batch_idx][query_idx];

  auto key_j = reinterpret_cast<vec_t*>(key[batch_idx][l].data());
  scalar_t attn_v = 0;
  scalar_t grad_attn_v = 0;

  for (int64_t k = 0; k < K / kVecSize; k += 1) {
    //sputnik::VectorCompute<vec_t>::Dot(query_block[k], key_j[k], &attn_v);
    //sputnik::VectorCompute<vec_t>::Dot(__ldg(qb + k), __ldg(kb + k), &attn_v);
    sputnik::VectorCompute<vec_t>::Dot(query_cache[threadIdx.x][k], key_cache[threadIdx.y][k], &attn_v);
    sputnik::VectorCompute<vec_t>::Dot(__ldg(gb + k), __ldg(vb + k), &grad_attn_v);
  }
  attn_v = std::exp(attn_v - normalizer);


  /*
  for (int64_t k = 0; k < K / kVecSize; k++) {
    sputnik::VectorCompute<vec_t>::Dot(__ldg(gb + k), __ldg(vb + k), &grad_attn_v);
    //sputnik::VectorCompute<vec_t>::Dot(grad_out_cache[threadIdx.x][k], value_cache[threadIdx.y][k], &grad_attn_v);
  }*/

  fact[threadIdx.x][threadIdx.y] =  attn_v * grad_attn_v - attn_v * tmp_sum;
  __syncthreads();



  for (int64_t k = threadIdx.y; k < K / kVecSize; k+= blockDim.y) {
      vec_t res = {0};
      for (int64_t i = 0; i < KS2; i++) {
        sputnik::VectorCompute<vec_t>::FMA(fact[threadIdx.x][i], key_cache[i][k], &res);
        //sputnik::VectorCompute<vec_t>::FMA(fact[threadIdx.x][i], __ldg(kb + i * K / kVecSize + k), &res);
      }
      //if (threadIdx.y == 0)
        myGpuAtomicAdd(&grad_q[batch_idx][query_idx][k * kVecSize], res);
  }

  for (int64_t k = threadIdx.x; k < K / kVecSize; k+= blockDim.x) {
      vec_t res = {0};
      for (int64_t i = 0; i < KS1; i++) {
        sputnik::VectorCompute<vec_t>::FMA(fact[i][threadIdx.y], query_cache[i][k], &res);
        //sputnik::VectorCompute<vec_t>::FMA(fact[i][threadIdx.y], __ldg(qb + i * K / kVecSize + k), &res);
      }
      //if (threadIdx.x == 0)
        myGpuAtomicAdd(&grad_k[batch_idx][l][k * kVecSize], res);
  }
/*
  for (int64_t k = 0; k < K / kVecSize; k++) {
      vec_t res = {0};
      scalar_t ttmp = attn_v * grad_attn_v - attn_v * tmp_sum;
      //sputnik::VectorCompute<vec_t>::FMA(ttmp, key_j[k], &res);
      sputnik::VectorCompute<vec_t>::FMA(ttmp, key_cache[threadIdx.y][k], &res);
      //sputnik::VectorCompute<vec_t>::FMA(ttmp, key_cache[threadIdx.y][k], &tmp_grad1[threadIdx.x][k]);
      //myGpuAtomicAdd(&grad_q[batch_idx][query_idx][k * kVecSize], res);
  }


  for (int64_t k = 0; k < K / kVecSize; k++) {
      //vec_t res = {0};
      scalar_t ttmp = attn_v * grad_attn_v - attn_v * tmp_sum;
      //sputnik::VectorCompute<vec_t>::FMA(ttmp, query_block[k], &res);
      sputnik::VectorCompute<vec_t>::FMA(ttmp, query_cache[threadIdx.x][k], &res);
      //sputnik::VectorCompute<vec_t>::FMA(ttmp, query_cache[threadIdx.x][k], &tmp_grad2[threadIdx.y][k]);
      //myGpuAtomicAdd(&grad_k[batch_idx][l][k * kVecSize], res);
  }
*/
/*
  if ((threadIdx.x == 0) && (threadIdx.y == 0)) {
    for (int64_t k = 0; k < K / kVecSize; k++) {
      vec_t res0 = {0};
      for (int i = 0; i < KS1; i++)
      {
        sputnik::VectorCompute<vec_t>::FMA(scalar_t(1), tmp_grad1[i][k], &res0);
      }
      myGpuAtomicAdd(&grad_q[batch_idx][query_idx][k * kVecSize], res0);

      vec_t res1 = {0};
      for (int i = 0; i < KS2; i++)
      {
        sputnik::VectorCompute<vec_t>::FMA(scalar_t(1), tmp_grad2[i][k], &res1);
      }
      myGpuAtomicAdd(&grad_k[batch_idx][l][k * kVecSize], res1);
    }
  }*/


}



std::tuple<at::Tensor, at::Tensor, at::Tensor> attention_backward(
    const at::Tensor& grad_out,
    const at::Tensor& query,
    const at::Tensor& key,
    const at::Tensor& value,
    const at::Tensor& logsumexp
    // const at::Tensor& mask
) {
  TORCH_CHECK(query.dim() == grad_out.dim());
  TORCH_CHECK(query.dim() == key.dim());
  TORCH_CHECK(query.dim() == value.dim());
  // TORCH_CHECK(query.dim() == mask.dim());
  TORCH_CHECK(query.dim() == 3);

  TORCH_CHECK(query.size(0) == grad_out.size(0));
  TORCH_CHECK(query.size(1) == grad_out.size(1));
  TORCH_CHECK(query.size(2) == grad_out.size(2));

  TORCH_CHECK(query.size(2) == key.size(2));
  TORCH_CHECK(query.size(0) == key.size(0));

  TORCH_CHECK(query.size(0) == value.size(0));
  TORCH_CHECK(key.size(1) == value.size(1));
  TORCH_CHECK(
      query.size(2) ==
      value.size(2)); // TODO: drop this limitation in the future

  TORCH_CHECK(query.is_cuda(), "query must be a CUDA tensor");
  TORCH_CHECK(key.is_cuda(), "key must be a CUDA tensor");
  TORCH_CHECK(value.is_cuda(), "value must be a CUDA tensor");
  TORCH_CHECK(grad_out.is_cuda(), "grad_out must be a CUDA tensor");

  TORCH_CHECK(!query.is_sparse(), "query must be a dense tensor");
  TORCH_CHECK(!key.is_sparse(), "key must be a dense tensor");
  TORCH_CHECK(!value.is_sparse(), "value must be a dense tensor");
  TORCH_CHECK(!grad_out.is_sparse(), "grad_out must be a dense tensor");

  at::cuda::CUDAGuard device_guard(query.device());

  int64_t B = query.size(0);
  int64_t M = query.size(1);
  int64_t N = key.size(1);
  int64_t K = query.size(2);

  //at::Tensor res = at::empty({B, M, K}, query.options());
  at::Tensor grad_q = at::zeros_like(query);
  at::Tensor grad_k = at::zeros_like(key);
  at::Tensor grad_v = at::zeros_like(value);

  at::Tensor tmp_sum_i = at::empty({B, M}, query.options());

  using scalar_t = float;
  using vec_t = float4;
  // using vec_t = float;

  constexpr int TILE_SIZE = 16 * 8;
  constexpr int kVecSize = sizeof(vec_t) / sizeof(scalar_t);

  constexpr int64_t BUFFER_SIZE = 32 / kVecSize;
  constexpr int64_t kBlockSizeQ = 16;
  constexpr int64_t kBlockSizeK = 4;

  dim3 grid(ceil_div(M, int64_t(TILE_SIZE)), B);
  dim3 block(32, TILE_SIZE / kBlockSizeQ);

  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  // AT_DISPATCH_FLOATING_TYPES(
  //    query.scalar_type(), "attention_backward_kernel", [&] {
  attention_backward_kernel<
      scalar_t,
      vec_t,
      kBlockSizeQ,
      kBlockSizeK,
      BUFFER_SIZE><<<grid, block, 0, stream>>>(
      grad_q.packed_accessor<scalar_t, 3>(),
      grad_k.packed_accessor<scalar_t, 3>(),
      grad_v.packed_accessor<scalar_t, 3>(),
      grad_out.packed_accessor<scalar_t, 3>(),
      query.packed_accessor<scalar_t, 3>(),
      key.packed_accessor<scalar_t, 3>(),
      value.packed_accessor<scalar_t, 3>(),
      tmp_sum_i.packed_accessor<scalar_t, 2>(),
      logsumexp.packed_accessor<scalar_t, 2>());
  //   });


  constexpr int TILE_SIZE2Q = 32 * 4;
  constexpr int TILE_SIZE2K = 32 * 8;

  constexpr int64_t kBlockSizeQ2 = 8;//2;
  constexpr int64_t kBlockSizeK2 = 16;//32;

  //dim3 grid2(ceil_div(M, int64_t(TILE_SIZE2Q)), ceil_div(N, int64_t(TILE_SIZE2K)), B);
  //dim3 block2(TILE_SIZE2Q / kBlockSizeQ2, TILE_SIZE2K / kBlockSizeK2);

  dim3 grid2(ceil_div(M, int64_t(16)), ceil_div(N, int64_t(16)), B);
  dim3 block2(16, 16);
  // TODO: try adding a blockDim.x to iterate over k

  attention_backward_kernel2<
      scalar_t,
      vec_t,
      kBlockSizeQ2,
      kBlockSizeK2,
      BUFFER_SIZE><<<grid2, block2, 0, stream>>>(
      grad_q.packed_accessor<scalar_t, 3>(),
      grad_k.packed_accessor<scalar_t, 3>(),
      grad_v.packed_accessor<scalar_t, 3>(),
      grad_out.packed_accessor<scalar_t, 3>(),
      query.packed_accessor<scalar_t, 3>(),
      key.packed_accessor<scalar_t, 3>(),
      value.packed_accessor<scalar_t, 3>(),
      tmp_sum_i.packed_accessor<scalar_t, 2>(),
      logsumexp.packed_accessor<scalar_t, 2>());
  //   });


  AT_CUDA_CHECK(hipGetLastError());

  return std::make_tuple(grad_q, grad_k, grad_v);
}

} // namespace

TORCH_LIBRARY_IMPL(xformers, CUDA, m) {
  m.impl(
      TORCH_SELECTIVE_NAME("xformers::efficient_attention"),
      TORCH_FN(attention));
  m.impl(
      TORCH_SELECTIVE_NAME("xformers::efficient_attention_backward"),
      TORCH_FN(attention_backward));
}
