#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <ATen/Parallel.h>
#include <torch/library.h>
#include <cmath>
#include <vector>


#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>


namespace {



#define CUDA_1D_KERNEL_LOOP(i, n)                                \
  for (int i = (blockIdx.x * blockDim.x) + threadIdx.x; i < (n); \
       i += (blockDim.x * gridDim.x))

template <typename integer>
constexpr __host__ __device__ inline integer ceil_div(integer n, integer m) {
  return (n + m - 1) / m;
}

template <typename scalar_t>
__global__ void attention_kernel(
    at::PackedTensorAccessor<scalar_t, 3> output,
    at::PackedTensorAccessor<scalar_t, 3> query,
    at::PackedTensorAccessor<scalar_t, 3> key,
    at::PackedTensorAccessor<scalar_t, 3> value
    ) {
  constexpr int64_t BLOCK = 32;
  int64_t K = query.size(2);
  int64_t B = query.size(0);
  int64_t M = query.size(1);
  int64_t N = key.size(1);

  int64_t i = blockIdx.y;
  //int64_t j = blockIdx.x;
  int64_t j = blockIdx.x * (blockDim.y * 2) + threadIdx.y * 2;

      {{
        //auto aar = query[i][j].data();
        auto aar = reinterpret_cast<float4 *>(query[i][j].data());
        auto aar2 = reinterpret_cast<float4 *>(query[i][j+1].data());

        //auto oo = output[i][j].data();
        auto oo = reinterpret_cast<float4 *>(output[i][j].data());
        auto oo2 = reinterpret_cast<float4 *>(output[i][j+1].data());
        scalar_t s_prime = 0;
        scalar_t m_prime = -std::numeric_limits<scalar_t>::infinity();

        scalar_t s_prime2 = 0;
        scalar_t m_prime2 = -std::numeric_limits<scalar_t>::infinity();
        //for (int64_t l = threadIdx.x * BLOCK; l < N; l+=BLOCK * blockDim.x) {
        for (int64_t l = 0; l < N; l+=BLOCK) {
          //auto bar = key[i][l].data();
          auto bar = reinterpret_cast<float4 *>(key[i][l].data());
          scalar_t si[BLOCK] = {0};
          scalar_t si2[BLOCK] = {0};
          //for (int64_t k = threadIdx.x; k < K; k+=32) {
          for (int64_t k = 0; k < K / 4; k+=1) {
            //auto aaar = aar[k];
            //auto aaar = __ldg(aar + k);
            float4 aaar = __ldg(aar + k);
            float4 aaar2 = __ldg(aar2 + k);
            for (int64_t rr = 0; rr < BLOCK; rr++) {
              float4 bbb = bar[k + K / 4 * rr];
              si[rr] += aaar.x * bbb.x + aaar.y * bbb.y + aaar.z * bbb.z + aaar.w * bbb.w;
              si2[rr] += aaar2.x * bbb.x + aaar2.y * bbb.y + aaar2.z * bbb.z + aaar2.w * bbb.w;

              //si[rr] += aaar * bar[k + K * rr];
              //si[rr] += aaar * __ldg(bar + k + K * rr);
            }
          }

          //for (int64_t rr = 0; rr < BLOCK; rr++) {
          //  for (int stride = 16; stride > 0; stride >>= 1) {
          //    si[rr] += __shfl_xor_sync(0xffffffff, si[rr], stride, 32);
          //  }
          //}

          scalar_t m_i = si[0] > m_prime ? si[0] : m_prime;
          for (int64_t rr = 1; rr < BLOCK; rr++) {
            m_i = si[rr] > m_i ? si[rr] : m_i;
          }

          scalar_t m_i2 = si2[0] > m_prime2 ? si2[0] : m_prime2;
          for (int64_t rr = 1; rr < BLOCK; rr++) {
            m_i2 = si2[rr] > m_i2 ? si2[rr] : m_i2;
          }
          //s_prime = m_i;  // TODO: only for testing, remove!!!

          //auto vi = value[i][l].data();
          auto vi = reinterpret_cast<float4 *>(value[i][l].data());

          scalar_t m_delta;
          scalar_t s_delta[BLOCK];
          m_delta = std::exp(m_prime - m_i);


          scalar_t m_delta2;
          scalar_t s_delta2[BLOCK];
          m_delta2 = std::exp(m_prime2 - m_i2);

          for (int64_t rr = 0; rr < BLOCK; rr++)
            s_delta[rr] = std::exp(si[rr] - m_i);

          for (int64_t rr = 0; rr < BLOCK; rr++)
            s_delta2[rr] = std::exp(si2[rr] - m_i2);

          //for (int64_t k = threadIdx.x; k < K; k+=blockDim.x) {
          for (int64_t k = 0; k < K/4; k+=1) {
            //oo[k] = oo[k] * m_delta;
            float4 tmp = oo[k];
            tmp.x = tmp.x * m_delta;
            tmp.y = tmp.y * m_delta;
            tmp.z = tmp.z * m_delta;
            tmp.w = tmp.w * m_delta;

            float4 tmp3 = oo2[k];
            tmp3.x = tmp3.x * m_delta2;
            tmp3.y = tmp3.y * m_delta2;
            tmp3.z = tmp3.z * m_delta2;
            tmp3.w = tmp3.w * m_delta2;
            for (int64_t rr = 0; rr < BLOCK; rr++) {
              //oo[k] += vi[k + K * rr] * s_delta[rr];
              float4 tmp2 = vi[k + K / 4 * rr];
              tmp.x += tmp2.x * s_delta[rr];
              tmp.y += tmp2.y * s_delta[rr];
              tmp.z += tmp2.z * s_delta[rr];
              tmp.w += tmp2.w * s_delta[rr];

              tmp3.x += tmp2.x * s_delta2[rr];
              tmp3.y += tmp2.y * s_delta2[rr];
              tmp3.z += tmp2.z * s_delta2[rr];
              tmp3.w += tmp2.w * s_delta2[rr];
              //oo[k] += __ldg(vi + k + K * rr) * s_delta[rr];
            }
            oo[k] = tmp;
            oo2[k] = tmp3;
          }
          s_prime = s_prime * m_delta;
          for (int64_t rr = 0; rr < BLOCK; rr++)
            s_prime += s_delta[rr];

          m_prime = m_i;


          s_prime2 = s_prime2 * m_delta2;
          for (int64_t rr = 0; rr < BLOCK; rr++)
            s_prime2 += s_delta2[rr];

          m_prime2 = m_i2;
        }

        //for (int64_t k = threadIdx.x; k < K; k+=blockDim.x) {
        for (int64_t k = 0; k < K / 4; k+=1) {
          //oo[k] /= s_prime;
          float4 tmp = oo[k];
          float4 tmp2 = oo2[k];
          tmp.x /= s_prime;
          tmp.y /= s_prime;
          tmp.z /= s_prime;
          tmp.w /= s_prime;

          tmp2.x /= s_prime2;
          tmp2.y /= s_prime2;
          tmp2.z /= s_prime2;
          tmp2.w /= s_prime2;
          oo[k] = tmp;
          oo2[k] = tmp2;
        }
      }
  }
}

at::Tensor attention(
    const at::Tensor& query,
    const at::Tensor& key,
    const at::Tensor& value
    //const at::Tensor& mask
    ) {
  TORCH_CHECK(query.dim() == key.dim());
  //TORCH_CHECK(query.dim() == mask.dim());
  TORCH_CHECK(query.dim() == 3);
  TORCH_CHECK(query.size(2) == key.size(2));
  TORCH_CHECK(query.size(0) == key.size(0));
  //TORCH_CHECK(query.size(1) == mask.size(1));
  //TORCH_CHECK(query.size(2) == mask.size(2));
  //TORCH_CHECK(query.size(0) == mask.size(0));

  /*
  TORCH_CHECK(!a.is_cuda(), "a must be a CPU tensor");
  TORCH_CHECK(!b.is_cuda(), "b must be a CPU tensor");
  TORCH_CHECK(!mask.is_cuda(), "mask must be a CPU tensor");

  TORCH_CHECK(!a.is_sparse(), "a must be a dense tensor");
  TORCH_CHECK(!b.is_sparse(), "b must be a dense tensor");
  //TORCH_CHECK(mask.is_sparse(), "mask must be a sparse tensor");
  */
  TORCH_CHECK(query.is_contiguous());
  TORCH_CHECK(key.is_contiguous());
  TORCH_CHECK(value.is_contiguous());

  at::cuda::CUDAGuard device_guard(query.device());

  int64_t B = query.size(0);
  int64_t M = query.size(1);
  int64_t N = key.size(1);
  int64_t K = query.size(2);


  at::Tensor res = at::zeros({B, M, K}, query.options());

  int64_t grain_size = 32; // TODO: tune this
  //at::Tensor buffer = at::empty({B, grain_size, K}, query.options());
  //at::Tensor buffer = at::empty({at::get_num_threads(), 1, K}, query.options());


  //dim3 grid(std::min(
  //    ceil_div(static_cast<int64_t>(B), static_cast<int64_t>(512)),
  //    static_cast<int64_t>(4096)));
  //dim3 block(512);
  //dim3 grid(M / 32, B);
  dim3 grid(M / 32, B);
  //dim3 block(32, 32);
  //dim3 block(4, 32);
  dim3 block(1, 16);
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  AT_DISPATCH_FLOATING_TYPES(
      query.scalar_type(), "attention_kernel", [&] {
        attention_kernel<scalar_t><<<grid, block, 0, stream>>>(
            res.packed_accessor<scalar_t, 3>(),
            query.packed_accessor<scalar_t, 3>(),
            key.packed_accessor<scalar_t, 3>(),
            value.packed_accessor<scalar_t, 3>()
            //buffer.accessor<scalar_t, 3>()
            //idxs.accessor<int64_t, 2>()
            );
      });

  return res;
}

} // namespace

TORCH_LIBRARY_IMPL(xformers, CUDA, m) {
  m.impl(
      TORCH_SELECTIVE_NAME("xformers::efficient_attention"),
      TORCH_FN(attention));
}
