#include <ATen/ATen.h>
#include <torch/types.h>

#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include <sputnik/spmm/cuda_spmm.h>

at::Tensor spmm_sputnik(
    const at::Tensor& b,
    const at::Tensor& row_indices,
    const at::Tensor& values,
    const at::Tensor& row_offsets,
    const at::Tensor& column_indices,
    int64_t m) {
  hipStream_t stream = at::cuda::getCurrentCUDAStream();
  int batch = b.size(0);
  int k = b.size(1);
  int n = b.size(2);

  int nonzeros = column_indices.size(0);
  TORCH_CHECK(
      batch == 1 || nonzeros % 4 == 0,
      "If batch size > 1 then number of nonzeros should be a multiple of 4");

  at::Tensor output = at::empty({batch, m, n}, b.options());

  for (int i = 0; i < batch; i++) {
    // TODO investigate misaligned address errors in values ptr
    AT_CUDA_CHECK(sputnik::CudaSpmm(
        m,
        k,
        n,
        nonzeros,
        row_indices.data_ptr<int>(),
        values.data_ptr<float>() + nonzeros * i,
        row_offsets.data_ptr<int>(),
        column_indices.data_ptr<int>(),
        b.data_ptr<float>() + k * n * i,
        output.data_ptr<float>() + m * n * i,
        stream));
  }

  return output;
}

TORCH_LIBRARY_FRAGMENT(xformers, m) {
  m.def(TORCH_SELECTIVE_SCHEMA(
      "xformers::spmm_sputnik(Tensor b, Tensor row_indices, Tensor values, Tensor row_offsets, Tensor column_indices, int m) -> Tensor"));
}

TORCH_LIBRARY_IMPL(xformers, CUDA, m) {
  m.impl(
      TORCH_SELECTIVE_NAME("xformers::spmm_sputnik"), TORCH_FN(spmm_sputnik));
}
