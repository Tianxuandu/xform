#include "hip/hip_runtime.h"
#include <ATen/ATen.h>
#include <torch/library.h>
#include <cmath>
#include <vector>

#include <hip/hip_fp16.h>

#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include "cutlass/gemm/gemm.h"
#include "cutlass/layout/matrix.h"
#include "cutlass/layout/vector.h"
#include "cutlass/numeric_types.h"

#include "attention_scaling_coefs_updater.h"
#include "cutlass/epilogue/threadblock/default_epilogue_simt.h"
#include "cutlass/epilogue/threadblock/default_epilogue_tensor_op.h"
#include "cutlass/epilogue/threadblock/default_epilogue_volta_tensor_op.h"
#include "cutlass/gemm/device/default_gemm_configuration.h"
#include "cutlass/gemm/kernel/default_gemm.h"
#include "cutlass/gemm/threadblock/default_mma.h"
#include "cutlass/gemm/threadblock/default_mma_core_simt.h"
#include "cutlass/gemm/threadblock/default_mma_core_sm70.h"
#include "cutlass/gemm/threadblock/default_mma_core_sm75.h"
#include "cutlass/gemm/threadblock/default_mma_core_sm80.h"
#include "cutlass/gemm/threadblock/threadblock_swizzle.h"
#include "cutlass/matrix_shape.h"
#include "cutlass/platform/platform.h"
#include "cutlass/transform/threadblock/predicated_tile_iterator.h"
#include "debug_utils.h"
#include "epilogue_rescale_output.h"
#include "find_default_mma.h"
#include "gemm_kernel_utils.h"
#include "mma_from_smem.h"

#include <inttypes.h>

using namespace gemm_kernel_utils;

namespace {
template <
    // The datatype of Q/K/V
    typename scalar_t_,
    // Intermediate accumulation type (including softmax)
    typename accum_t_,
    // Output type (only float tested so far)
    typename output_t_,
    // If Q/K/V are correctly aligned in memory and we can run a fast kernel
    bool isAligned_>
struct AttentionKernelInfo {
  using scalar_t = scalar_t_;
  using accum_t = accum_t_;
  using output_t = output_t_;
  using lse_scalar_t = float;
  static constexpr bool kIsAligned = isAligned_;
  static constexpr int32_t kAlignLSE = 32; // block size of backward

  // Blocks
  // NOTE: Looks like 16 works better for K <= 64
  static constexpr int64_t kQueriesPerBlock = 64;
  static constexpr int64_t kNumWarpsPerBlock = 4;
  static constexpr int64_t kWarpSize = 32;

  struct Params {
    // Input tensors
    scalar_t* query_ptr; // [num_queries, head_dim]
    scalar_t* key_ptr; // [num_keys, head_dim]
    scalar_t* value_ptr; // [num_keys, head_dim_value]

    // Output tensors
    output_t* output_ptr; // [num_queries, head_dim_value]
    lse_scalar_t* logsumexp_ptr; // [num_queries] - can be 0

    // Dimensions/strides
    int32_t head_dim;
    int32_t head_dim_value;
    int32_t num_queries;
    int32_t num_keys;
    int32_t num_batches;

    __device__ void advance_batches(int32_t batch_id) {
      auto lse_dim = ceil_div((int32_t)num_queries, kAlignLSE) * kAlignLSE;

      query_ptr += batch_id * head_dim * num_queries;
      key_ptr += batch_id * head_dim * num_keys;
      value_ptr += batch_id * head_dim_value * num_keys;
      output_ptr += batch_id * head_dim_value * num_queries;
      if (logsumexp_ptr != nullptr) {
        logsumexp_ptr += batch_id * lse_dim;
      }
    }

    __host__ dim3 getBlocksGrid() const {
      return dim3(
          1, ceil_div(num_queries, (int32_t)kQueriesPerBlock), num_batches);
    }
    __host__ dim3 getThreadsGrid() const {
      return dim3(kWarpSize, kNumWarpsPerBlock, 1);
    }
  };
};

template <typename KernelInfo, typename ArchTag>
struct AttentionKernel {
  using scalar_t = typename KernelInfo::scalar_t;
  using accum_t = typename KernelInfo::accum_t;
  using output_t = typename KernelInfo::output_t;
  using Params = typename KernelInfo::Params;
  static constexpr bool kIsAligned = KernelInfo::kIsAligned;
  static constexpr int64_t kQueriesPerBlock = KernelInfo::kQueriesPerBlock;
  static constexpr int64_t kKeysPerBlock = 64;
  static constexpr int64_t kNumWarpsPerBlock = KernelInfo::kNumWarpsPerBlock;
  static constexpr int64_t kWarpSize = KernelInfo::kWarpSize;

  struct MM0 {
    /*
      In this first matmul, we compute a block of `Q @ K.T`.
      While the calculation result is still hot in registers, we update
      `mi`, `m_prime`, `s_prime` in shared-memory, and then store this value
      into a shared-memory ("AccumulatorSharedStorage") that is used later as
      operand A for the second matmul (see MM1)
    */
    using GemmType = DefaultGemmType<ArchTag, scalar_t>;

    using OpClass = typename GemmType::OpClass;
    using DefaultConfig =
        typename cutlass::gemm::device::DefaultGemmConfiguration<
            OpClass,
            ArchTag,
            scalar_t,
            scalar_t,
            scalar_t, // ElementC
            accum_t // ElementAccumulator
            >;
    static constexpr int64_t kAlignmentA =
        kIsAligned ? DefaultConfig::kAlignmentA : GemmType::kMinimumAlignment;
    static constexpr int64_t kAlignmentB =
        kIsAligned ? DefaultConfig::kAlignmentB : GemmType::kMinimumAlignment;
    using ThreadblockShape = cutlass::gemm::
        GemmShape<kQueriesPerBlock, kKeysPerBlock, GemmType::ThreadK>;
    using WarpShape = cutlass::gemm::GemmShape<32, 32, GemmType::WarpK>;
    using DefaultMma = typename cutlass::gemm::threadblock::FindDefaultMma<
        scalar_t, // ElementA,
        cutlass::layout::RowMajor, // LayoutA,
        kAlignmentA,
        scalar_t, // ElementB,
        cutlass::layout::ColumnMajor, // LayoutB,
        kAlignmentB,
        accum_t,
        cutlass::layout::RowMajor, // LayoutC,
        OpClass,
        ArchTag, // ArchTag
        ThreadblockShape, // ThreadblockShape
        WarpShape, // WarpShape
        typename GemmType::InstructionShape, // InstructionShape
        DefaultConfig::kStages, // Should use `DefaultConfig::kStages`, but that
                                // uses too much smem
        typename GemmType::Operator // Operator
        >::DefaultMma;
    using MmaCore = typename DefaultMma::MmaCore;
    using IteratorA = typename DefaultMma::IteratorA;
    using IteratorB = typename DefaultMma::IteratorB;
    using Mma = typename DefaultMma::ThreadblockMma;
    using ScalingCoefsUpdater = typename DefaultAttentionScalingCoefsUpdater<
        typename Mma::Operator::IteratorC,
        accum_t,
        kWarpSize>::Updater;

    // Epilogue to store to shared-memory in a format that we can use later for
    // the second matmul
    using B2bGemm = typename cutlass::gemm::threadblock::B2bGemm<
        typename Mma::Operator::IteratorC,
        typename Mma::Operator,
        scalar_t,
        WarpShape,
        ThreadblockShape>;
    using AccumulatorSharedStorage = typename B2bGemm::AccumulatorSharedStorage;
  };

  struct MM1 {
    /**
      Second matmul: perform `attn @ V` where `attn` is the attention (not
      normalized) and stored in shared memory
    */
    using GemmType = DefaultGemmType<ArchTag, scalar_t>;

    using OpClass = typename GemmType::OpClass;
    using DefaultConfig =
        typename cutlass::gemm::device::DefaultGemmConfiguration<
            OpClass,
            ArchTag,
            scalar_t,
            scalar_t,
            output_t, // ElementC
            accum_t // ElementAccumulator
            >;
    static constexpr int64_t kAlignmentA =
        DefaultConfig::kAlignmentA; // from smem
    static constexpr int64_t kAlignmentB =
        kIsAligned ? DefaultConfig::kAlignmentB : GemmType::kMinimumAlignment;
    using ThreadblockShape = cutlass::gemm::
        GemmShape<kQueriesPerBlock, kKeysPerBlock, GemmType::ThreadK>;
    using WarpShape = cutlass::gemm::GemmShape<32, 32, GemmType::WarpK>;
    using InstructionShape = typename GemmType::InstructionShape;

    using LayoutB = cutlass::layout::RowMajor;
    using DefaultGemm = cutlass::gemm::kernel::DefaultGemm<
        scalar_t, // ElementA,
        cutlass::layout::RowMajor, // LayoutA,
        kAlignmentA,
        scalar_t, // ElementB,
        LayoutB, // LayoutB,
        kAlignmentB,
        output_t,
        cutlass::layout::RowMajor, // LayoutC,
        accum_t,
        OpClass,
        ArchTag,
        ThreadblockShape,
        WarpShape,
        typename GemmType::InstructionShape,
        typename DefaultConfig::EpilogueOutputOp,
        void, // ThreadblockSwizzle - not used
        DefaultConfig::kStages,
        false, // SplitKSerial
        typename GemmType::Operator>;

    using DefaultMmaFromSmem =
        typename cutlass::gemm::threadblock::DefaultMmaFromSharedMemory<
            typename DefaultGemm::Mma,
            typename MM0::AccumulatorSharedStorage>;
    using Mma = typename DefaultMmaFromSmem::Mma;
    using IteratorB = typename Mma::IteratorB;
    using WarpCount = typename Mma::WarpCount;

    using DefaultEpilogue = typename DefaultGemm::Epilogue;
    using OutputTileIterator = typename DefaultEpilogue::OutputTileIterator;

    struct SharedStorageMM1 {
      union {
        // Storing parts of `V` during the matmul
        typename Mma::SharedStorage mm;
        // Used by the Epilogue (so we can reuse the same memory space)
        typename DefaultEpilogue::SharedStorage epilogue;
      };
    };

    static __device__ void compute_dot_product_att_value(
        Params const& p,
        SharedStorageMM1& shared_storage_mm,
        typename MM0::AccumulatorSharedStorage& shared_storage_si,
        int32_t const& iter_key_start,
        cutlass::Array<accum_t, kQueriesPerBlock> const& m_prime,
        cutlass::Array<accum_t, kQueriesPerBlock> const& s_prime,
        bool isLast) {
      cutlass::gemm::GemmCoord problem_size(
          std::min(
              (int32_t)kQueriesPerBlock, p.num_queries - query_start()), // M
          p.head_dim_value, // N
          std::min(
              int32_t(kKeysPerBlock),
              p.num_keys - iter_key_start) // K
      );

      typename IteratorB::Params params_B(LayoutB(p.head_dim_value));

      static_assert(
          WarpCount::kM * WarpCount::kN * WarpCount::kK == kNumWarpsPerBlock);

      const int64_t nBlockN =
          ceil_div((int64_t)problem_size.n(), int64_t(ThreadblockShape::kN));
      for (int blockN = 0; blockN < nBlockN; ++blockN) {
        /*
        Run the matmul `attn @ V` for a block of attn and V.
        `attn` is read from shared memory (in `shared_storage_si`)
        `V` is read from global memory (with iterator_B)
        */
        cutlass::gemm::GemmCoord tb_tile_offset = {0, blockN, 0};

        cutlass::MatrixCoord tb_offset_B{
            tb_tile_offset.k(), tb_tile_offset.n() * Mma::Shape::kN};

        typename Mma::IteratorB iterator_B(
            params_B,
            p.value_ptr + iter_key_start * p.head_dim_value,
            {problem_size.k(), problem_size.n()},
            thread_id(),
            tb_offset_B);

        typename Mma::FragmentC accum;
        accum.clear();

        Mma mma(
            shared_storage_mm.mm,
            shared_storage_si,
            thread_id(),
            warp_id(),
            lane_id(),
            problem_size.k());

        int gemm_k_iterations =
            (problem_size.k() + Mma::Shape::kK - 1) / Mma::Shape::kK;

        // Compute threadblock-scoped matrix multiply-add and store it in accum
        // (in registers)
        __syncthreads(); // we share shmem between mma and epilogue
        mma(gemm_k_iterations, accum, iterator_B, accum);
        __syncthreads(); // we share shmem between mma and epilogue

        /*
          Epilogue: Store the following into global memory
          output <- alpha * accumulator + beta * source
            with:
              alpha = 1 / s_prime (to normalize when isLast=True, 1 otherwise)
              beta = alpha / m_prime (renormalize the output when the max
          changes) source is the current output
        */
        int col = blockN * Mma::Shape::kN;
        OutputTileIterator output_tile_it(
            typename OutputTileIterator::Params{(int32_t)p.head_dim_value},
            p.output_ptr + query_start() * p.head_dim_value + col,
            typename OutputTileIterator::TensorCoord{
                p.num_queries - query_start(), p.head_dim_value - col},
            thread_id());
        OutputTileIterator source_tile_it(
            typename OutputTileIterator::Params{(int32_t)p.head_dim_value},
            p.output_ptr + query_start() * p.head_dim_value + col,
            typename OutputTileIterator::TensorCoord{
                p.num_queries - query_start(), p.head_dim_value - col},
            thread_id());
        using ElementCompute =
            typename DefaultConfig::EpilogueOutputOp::ElementCompute;
        DISPATCH_BOOL(
            iter_key_start == 0, kIsFirst, ([&]() {
              DISPATCH_BOOL(
                  isLast, kIsLast, ([&]() {
                    using EpilogueOutputOp = typename cutlass::epilogue::
                        thread::MemoryEfficientAttentionNormalize<
                            output_t,
                            DefaultConfig::EpilogueOutputOp::kCount,
                            typename DefaultConfig::EpilogueOutputOp::
                                ElementAccumulator,
                            typename DefaultConfig::EpilogueOutputOp::
                                ElementCompute,
                            kIsFirst,
                            kIsLast,
                            cutlass::Array<ElementCompute, kQueriesPerBlock>>;
                    using Epilogue = typename cutlass::epilogue::threadblock::
                        EpilogueWithRowId<
                            typename DefaultEpilogue::Shape,
                            typename Mma::Operator,
                            DefaultEpilogue::kPartitionsK,
                            typename DefaultEpilogue::OutputTileIterator,
                            typename DefaultEpilogue::
                                AccumulatorFragmentIterator,
                            typename DefaultEpilogue::WarpTileIterator,
                            typename DefaultEpilogue::SharedLoadIterator,
                            EpilogueOutputOp,
                            typename DefaultEpilogue::Padding,
                            DefaultEpilogue::kFragmentsPerIteration,
                            true // IterationsUnroll
                            >;
                    EpilogueOutputOp rescale(s_prime, m_prime);
                    Epilogue epilogue(
                        shared_storage_mm.epilogue,
                        thread_id(),
                        warp_id(),
                        lane_id());
                    epilogue(rescale, output_tile_it, accum, source_tile_it);
                  }));
            }));
      }
    }
  };

  static constexpr int64_t kAlignmentQ = MM0::kAlignmentA;
  static constexpr int64_t kAlignmentK = MM0::kAlignmentB;
  static constexpr int64_t kAlignmentV = 1;

  struct SharedStorageAfterMM0 {
    // Everything here might be overwritten during MM0
    typename MM0::AccumulatorSharedStorage si;
    cutlass::Array<accum_t, kQueriesPerBlock> mi;
    typename MM1::SharedStorageMM1 mm1;
  };

  struct SharedStorage {
    cutlass::Array<accum_t, kQueriesPerBlock> m_prime;
    cutlass::Array<accum_t, kQueriesPerBlock> s_prime;
    union {
      typename MM0::Mma::SharedStorage mm0;
      SharedStorageAfterMM0 after_mm0;
    };
  };

  static void __device__ attention_kernel(Params const& p) {
    int8_t lane_id = threadIdx.x;
    int8_t warp_id = threadIdx.y;

    // In this block, we will only ever:
    // - read query[query_start:query_end, :]
    // - write to output[query_start:query_end, :]

    extern __shared__ char smem_buffer[];
    SharedStorage& shared_storage = *((SharedStorage*)smem_buffer);
    auto& m_prime = shared_storage.m_prime;
    auto& s_prime = shared_storage.s_prime;
    auto& si = shared_storage.after_mm0.si;
    auto& mi = shared_storage.after_mm0.mi;

    static_assert(kQueriesPerBlock < kNumWarpsPerBlock * kWarpSize);
    if (thread_id() < kQueriesPerBlock) {
      s_prime[thread_id()] = accum_t(0);
      m_prime[thread_id()] = -std::numeric_limits<accum_t>::infinity();
    }

    // Iterate through keys
    for (int32_t iter_key_start = 0; iter_key_start < p.num_keys;
         iter_key_start += kKeysPerBlock) {
      __syncthreads(); // Need to have shared memory initialized, and `m_prime`
                       // updated from end of prev iter
      // 1. Compute dot-product into shared memory for each query
      // also calculates `mi`, and updates `m_prime` / `s_prime`
      compute_dot_product_qk(
          p, iter_key_start, m_prime, s_prime, shared_storage);

      __syncthreads();
      bool isLast = (iter_key_start + kKeysPerBlock) >= p.num_keys;

      // 4. Partial matmul with the values we have and V
      // `v* <- v* . exp(m* - mi) + v_i . exp(si - mi)`
      MM1::compute_dot_product_att_value(
          p,
          shared_storage.after_mm0.mm1,
          shared_storage.after_mm0.si,
          iter_key_start,
          m_prime,
          s_prime,
          isLast // 6. Divide by s_prime all of the values on the last
                 // iteration
      );
      __syncthreads(); // we modify `m_prime` after

      // 5. `m_prime` <- `mi` (`mi` will be overwritten during MM0)
      static_assert(kQueriesPerBlock < kNumWarpsPerBlock * kWarpSize);
      if (thread_id() < kQueriesPerBlock) {
        m_prime[thread_id()] = mi[thread_id()];
      }
      __syncthreads();
    }

    // 7. Calculate logsumexp
    // To make the backward easier, we pad logsumexp with `inf`
    // this avoids a few bound checks, and is not more expensive during fwd
    static_assert(kQueriesPerBlock < kNumWarpsPerBlock * kWarpSize);
    if (p.logsumexp_ptr && thread_id() < kQueriesPerBlock) {
      if (query_start() + thread_id() < p.num_queries) {
        p.logsumexp_ptr[query_start() + thread_id()] =
            accum_t(m_prime[thread_id()]) +
            std::log(accum_t(s_prime[thread_id()]));
      } else if (thread_id() < KernelInfo::kAlignLSE) {
        p.logsumexp_ptr[query_start() + thread_id()] =
            std::numeric_limits<accum_t>::infinity();
      }
    }
  }

  static __device__ void compute_dot_product_qk(
      Params const& p,
      int32_t const& iter_key_start,
      cutlass::Array<accum_t, kQueriesPerBlock>& m_prime,
      cutlass::Array<accum_t, kQueriesPerBlock>& s_prime,
      SharedStorage& shared_storage) {
    /*
    Computes the block-matrix product of:
    (a) query[query_start:query_end, :]
    with
    (b) key[iter_key_start:iter_key_start + kKeysPerBlock]
    and stores that into `shared_storage.si`
    */
    using MmaCore = typename MM0::MmaCore;
    using Mma = typename MM0::Mma;
    using IteratorA = typename MM0::IteratorA;
    using IteratorB = typename MM0::IteratorB;

    cutlass::gemm::GemmCoord problem_size(
        std::min((int32_t)kQueriesPerBlock, p.num_queries - query_start()),
        std::min(int32_t(kKeysPerBlock), p.num_keys - iter_key_start),
        p.head_dim);

    static_assert(
        MmaCore::WarpCount::kM * MmaCore::WarpCount::kN *
            MmaCore::WarpCount::kK ==
        kNumWarpsPerBlock);

    // Compute threadblock location
    cutlass::gemm::GemmCoord tb_tile_offset = {0, 0, 0};

    cutlass::MatrixCoord tb_offset_A{
        tb_tile_offset.m() * Mma::Shape::kM, tb_tile_offset.k()};

    cutlass::MatrixCoord tb_offset_B{
        tb_tile_offset.k(), tb_tile_offset.n() * Mma::Shape::kN};

    // Construct iterators to A and B operands
    typename Mma::IteratorA iterator_A(
        typename IteratorA::Params(typename MmaCore::LayoutA(p.head_dim)),
        p.query_ptr + query_start() * p.head_dim,
        {problem_size.m(), problem_size.k()},
        thread_id(),
        tb_offset_A);

    typename Mma::IteratorB iterator_B(
        typename IteratorB::Params(typename MmaCore::LayoutB(p.head_dim)),
        p.key_ptr + iter_key_start * p.head_dim,
        {problem_size.k(), problem_size.n()},
        thread_id(),
        tb_offset_B);

    auto my_warp_id = warp_id();
    auto my_lane_id = lane_id();

    // Construct thread-scoped matrix multiply
    Mma mma(shared_storage.mm0, thread_id(), my_warp_id, my_lane_id);

    typename Mma::FragmentC accum;

    accum.clear();

    auto gemm_k_iterations =
        (problem_size.k() + Mma::Shape::kK - 1) / Mma::Shape::kK;

    // Compute threadblock-scoped matrix multiply-add
    mma(gemm_k_iterations, accum, iterator_A, iterator_B, accum);
    __syncthreads();
    auto& mi = shared_storage.after_mm0.mi;
    static_assert(kQueriesPerBlock < kNumWarpsPerBlock * kWarpSize);
    if (thread_id() < kQueriesPerBlock) {
      mi[thread_id()] = m_prime[thread_id()];
    }
    __syncthreads();

    // Scale
    accum_t scale = accum_t(1.0 / std::sqrt(float(p.head_dim)));
    accum = cutlass::multiplies<typename Mma::FragmentC>()(scale, accum);

    typename Mma::Operator::IteratorC::TensorCoord iteratorC_tile_offset = {
        (tb_tile_offset.m() * Mma::WarpCount::kM) +
            (my_warp_id % Mma::WarpCount::kM),
        (tb_tile_offset.n() * Mma::WarpCount::kN) +
            (my_warp_id / Mma::WarpCount::kM)};
    // Update `mi` from accum stored in registers
    MM0::ScalingCoefsUpdater::update<kQueriesPerBlock>(
        accum,
        mi,
        m_prime,
        s_prime,
        lane_id(),
        thread_id(),
        warp_id(),
        p.num_keys - iter_key_start,
        iteratorC_tile_offset);

    // Output results to shared-memory
    int warp_idx_mn_0 = my_warp_id %
        (MM0::Mma::Base::WarpCount::kM * MM0::Mma::Base::WarpCount::kN);
    auto output_tile_coords = cutlass::MatrixCoord{
        warp_idx_mn_0 % MM0::Mma::Base::WarpCount::kM,
        warp_idx_mn_0 / MM0::Mma::Base::WarpCount::kM};

    MM0::B2bGemm::accumToSmem(
        shared_storage.after_mm0.si, accum, my_lane_id, output_tile_coords);
  }

  static __device__ __forceinline__ int8_t lane_id() {
    return threadIdx.x;
  }
  static __device__ __forceinline__ int8_t warp_id() {
    return threadIdx.y;
  }
  static __device__ __forceinline__ int16_t thread_id() {
    return threadIdx.x + threadIdx.y * blockDim.x;
  }
  static __device__ __forceinline__ int32_t query_start() {
    return blockIdx.y * kQueriesPerBlock;
  }
};

template <typename AKInfo>
__global__ void __launch_bounds__(
    // maxThreadsPerBlock specifies the maximum number of threads per block with
    // which the application will ever launch
    AKInfo::kWarpSize* AKInfo::kNumWarpsPerBlock,
    // minBlocksPerMultiprocessor is optional and specifies the desired minimum
    // number of resident blocks per multiprocessor
    // TODO: We get slightly better performance by *removing* this on A100
    12 / AKInfo::kNumWarpsPerBlock)
    attention_kernel_batched(typename AKInfo::Params p) {
#ifndef __CUDA_ARCH__
  using CurrentArch = cutlass::arch::Sm80;
#elif (__CUDA_ARCH__ >= 800)
  using CurrentArch = cutlass::arch::Sm80;
#elif (__CUDA_ARCH__ >= 750)
  using CurrentArch = cutlass::arch::Sm75;
#elif (__CUDA_ARCH__ >= 700)
  using CurrentArch = cutlass::arch::Sm70;
#elif (__CUDA_ARCH__ >= 500)
  using CurrentArch = cutlass::arch::Sm50;
#else
#error "Unsupported architecture in __CUDA_ARCH__"
#endif

#ifdef __CUDA_ARCH__
  static_assert(CurrentArch::kMinComputeCapability * 10 <= __CUDA_ARCH__);
#endif
  auto batch_id = blockIdx.z;
  p.advance_batches(batch_id);
  AttentionKernel<AKInfo, CurrentArch>::attention_kernel(p);
}

std::tuple<at::Tensor, at::Tensor, int64_t, int64_t>
efficient_attention_forward_generic(
    const at::Tensor& query,
    const at::Tensor& key,
    const at::Tensor& value,
    bool compute_logsumexp,
    const c10::optional<at::Tensor>& attn_bias_,
    double p) {
  TORCH_CHECK(p == 0.0, "Dropout is not supported at the moment");
  TORCH_CHECK(
      !attn_bias_.has_value(), "attn_bias is not supported at the moment");

  TORCH_CHECK(query.dim() == 3);
  TORCH_CHECK(key.dim() == 3);
  TORCH_CHECK(value.dim() == 3);

  TORCH_CHECK(query.size(2) == key.size(2));
  TORCH_CHECK(query.size(0) == key.size(0));

  TORCH_CHECK(query.is_cuda(), "query must be a CUDA tensor");
  TORCH_CHECK(key.is_cuda(), "key must be a CUDA tensor");
  TORCH_CHECK(value.is_cuda(), "value must be a CUDA tensor");

  TORCH_CHECK(!query.is_sparse(), "query must be a dense tensor");
  TORCH_CHECK(!key.is_sparse(), "key must be a dense tensor");
  TORCH_CHECK(!value.is_sparse(), "value must be a dense tensor");

  TORCH_CHECK(query.is_contiguous());
  TORCH_CHECK(key.is_contiguous());
  TORCH_CHECK(value.is_contiguous());

  at::cuda::CUDAGuard device_guard(query.device());
  hipStream_t stream = at::cuda::getCurrentCUDAStream();

  int64_t B = query.size(0);
  int64_t M = query.size(1);
  int64_t N = key.size(1);
  int64_t K = query.size(2);

  using accum_t = float;

  at::Tensor res;
  at::Tensor logsumexp;

  hipDeviceProp_t* properties =
      at::cuda::getDeviceProperties(query.device().index());
  const int computeCapability = properties->major * 10 + properties->minor;

#define DISPATCH_ARCHTAG(func)                                            \
  {                                                                       \
    if (computeCapability >= 80) {                                        \
      using ArchTag = cutlass::arch::Sm80;                                \
      func();                                                             \
    } else if (computeCapability >= 75) {                                 \
      using ArchTag = cutlass::arch::Sm75;                                \
      func();                                                             \
    } else if (computeCapability >= 70) {                                 \
      using ArchTag = cutlass::arch::Sm70;                                \
      func();                                                             \
    } else if (computeCapability >= 50) {                                 \
      using ArchTag = cutlass::arch::Sm50;                                \
      func();                                                             \
    } else {                                                              \
      TORCH_CHECK(                                                        \
          false,                                                          \
          "Your device is too old. We require compute capability >= 50"); \
    }                                                                     \
  }
// Dispatch to the right kernel
#define DISPATCH_TYPES(func)                                          \
  {                                                                   \
    if (query.scalar_type() == at::ScalarType::Float) {               \
      using scalar_t = float;                                         \
      using output_t = float;                                         \
      func();                                                         \
    } else if (query.scalar_type() == at::ScalarType::Half) {         \
      using scalar_t = cutlass::half_t;                               \
      using output_t = float;                                         \
      func();                                                         \
    } else {                                                          \
      TORCH_CHECK(false, "Only fp32 & half supported at the moment"); \
    }                                                                 \
  }

  DISPATCH_TYPES(([&]() {
    // Run a more efficient kernel (with `isAligned=True`) if memory is
    // correctly aligned
    using AlignedAKI = AttentionKernelInfo<scalar_t, accum_t, output_t, true>;
    bool isAligned;
    DISPATCH_ARCHTAG(([&]() {
      using AlignedAK = AttentionKernel<AlignedAKI, ArchTag>;
      isAligned =
          (query.stride(1) % AlignedAK::kAlignmentQ == 0 &&
           key.stride(1) % AlignedAK::kAlignmentK == 0 &&
           value.stride(1) % AlignedAK::kAlignmentV == 0);
      // TODO: Should we warn or log somewhere when we use a less efficient
      // kernel due to wrong alignment?
    }));
    DISPATCH_BOOL(
        isAligned, kIsAligned, ([&]() {
          using AKI =
              AttentionKernelInfo<scalar_t, accum_t, output_t, kIsAligned>;
          size_t smem_bytes = 0;
          DISPATCH_ARCHTAG(([&]() {
            using AK = AttentionKernel<AKI, ArchTag>;
            smem_bytes = sizeof(typename AK::SharedStorage);
            // Might happen on Sm80/half, where the minimum alignment is 32bits
            TORCH_CHECK(
                query.stride(1) % AK::kAlignmentQ == 0,
                "query is not correctly aligned");
            TORCH_CHECK(
                key.stride(1) % AK::kAlignmentK == 0,
                "key is not correctly aligned");
            TORCH_CHECK(
                value.stride(1) % AK::kAlignmentV == 0,
                "value is not correctly aligned");
          }));
          TORCH_INTERNAL_ASSERT(smem_bytes > 0, "No kernel found!?");

          res = at::zeros(
              {B, M, K},
              query.options().dtype(TypeTraits<output_t>::atScalarType()));
          // NOTE: Should be aligned (by padding) in case M is not a good number
          // for loading during backward
          constexpr decltype(M) kAlignLSE = 32; // block size of backward
          logsumexp = at::empty(
              {B, compute_logsumexp ? ceil_div(M, kAlignLSE) * kAlignLSE : 0},
              query.options().dtype(at::ScalarType::Float));

          constexpr auto kernel_fn = attention_kernel_batched<AKI>;
          if (smem_bytes > 0xc000) {
            TORCH_INTERNAL_ASSERT(
                computeCapability >= 70,
                "This kernel requires too much shared memory on this machine!");
            AT_CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(
                kernel_fn),
                hipFuncAttributeMaxDynamicSharedMemorySize,
                smem_bytes));
          }

          using m = TypeTraits<scalar_t>;
          typename AKI::Params p;
          p.query_ptr = (scalar_t*)query.data_ptr();
          p.key_ptr = (scalar_t*)key.data_ptr();
          p.value_ptr = (scalar_t*)value.data_ptr();
          p.logsumexp_ptr = compute_logsumexp
              ? (typename AKI::lse_scalar_t*)logsumexp.data_ptr()
              : nullptr;
          p.output_ptr = (output_t*)res.data_ptr();
          p.head_dim = query.size(2);
          p.head_dim_value = value.size(2);
          p.num_queries = query.size(1);
          p.num_keys = key.size(1);
          p.num_batches = B;
          kernel_fn<<<p.getBlocksGrid(), p.getThreadsGrid(), smem_bytes>>>(p);
        }));
  }));

  AT_CUDA_CHECK(hipGetLastError());
  return std::make_tuple(res, logsumexp, int64_t(), int64_t());
}
} // namespace

TORCH_LIBRARY_IMPL(xformers, CUDA, m) {
  m.impl(
      TORCH_SELECTIVE_NAME("xformers::efficient_attention_forward_generic"),
      TORCH_FN(efficient_attention_forward_generic));
}
